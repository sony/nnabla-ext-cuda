// Copyright (c) 2017 Sony Corporation. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

// convolution_cudnn.cu

#include <nbla/array.hpp>
#include <nbla/cuda/array/cuda_array.hpp>
#include <nbla/cuda/common.hpp>
#include <nbla/cuda/cudnn/cudnn.hpp>
#include <nbla/cuda/cudnn/function/convolution.hpp>
#include <nbla/variable.hpp>

namespace nbla {

template <typename T> void ConvolutionCudaCudnn<T>::wait_default_on_dgrad() {
  NBLA_CUDA_CHECK(hipEventRecord(*(this->default_event_), 0));
  NBLA_CUDA_CHECK(
      hipStreamWaitEvent(*(this->dgrad_stream_), *(this->default_event_), 0));
}

template <typename T> void ConvolutionCudaCudnn<T>::wait_dgrad_on_default() {
  NBLA_CUDA_CHECK(
      hipEventRecord(*(this->dgrad_event_), *(this->dgrad_stream_)));
  NBLA_CUDA_CHECK(hipStreamWaitEvent(0, *(this->dgrad_event_), 0));
}

template <typename T>
void ConvolutionCudaCudnn<T>::setup_impl(const Variables &inputs,
                                         const Variables &outputs) {
#if CUDNN_VERSION < 7100
  NBLA_CHECK(!this->channel_last_, error_code::value,
             "The passed argument channel_last_=true is not supported in this "
             "CUDNN version (%d).",
             (int)CUDNN_VERSION);
#endif
  cuda_set_device(device_);
  Convolution<T>::setup_impl(inputs, outputs);
  cudnn_handle_ = SingletonManager::get<CudnnHandleManager>()->handle(device_);

  auto event_deleter = [](hipEvent_t *ptr) {
    NBLA_CUDA_CHECK(hipEventDestroy(*ptr));
    std::default_delete<hipEvent_t>()(ptr);
  };

  dgrad_event_ = shared_ptr<hipEvent_t>(new hipEvent_t(), event_deleter);
  NBLA_CUDA_CHECK(
      hipEventCreateWithFlags(dgrad_event_.get(), hipEventDisableTiming));

  default_event_ = shared_ptr<hipEvent_t>(new hipEvent_t(), event_deleter);
  NBLA_CUDA_CHECK(
      hipEventCreateWithFlags(default_event_.get(), hipEventDisableTiming));

  dgrad_stream_ = SingletonManager::get<Cuda>()->get_stream(
      hipStreamNonBlocking, nbla::CudaStreamId::CONVOLUTION_BWD, device_);
  dgrad_handle_ = SingletonManager::get<CudnnHandleManager>()->handle(
      device_, *dgrad_stream_);

#if CUDNN_VERSION < 7000
  x_offset_ = this->inner_size_i_ / this->group_;
  y_offset_ = this->inner_size_o_ / this->group_;
  w_offset_ = this->channels_o_ * this->inner_size_k_ / this->group_;
  if (inputs.size() == 3) {
    b_offset_ = this->channels_o_ / this->group_;
  }
#endif

  // Create or query a resource.
  CudnnConvDesc desc{(int)this->kernel_.size(),
                     device_,
                     cudnn_data_type<T>::type(),
                     HIPDNN_CROSS_CORRELATION,
                     this->outer_size_,
                     this->channels_i_,
                     this->channels_o_,
                     this->group_,
                     this->channel_last_,
                     this->spatial_shape_i_,
                     this->kernel_,
                     this->pad_,
                     this->stride_,
                     this->dilation_};

  auto &rsc = SingletonManager::get<CudnnHandleManager>()->conv_resource;
  auto it = rsc.find(desc);
  if (it != rsc.end()) {
    // Found a previously created one.
    // std::cout << "Found previously created one: " << desc << std::endl;
    rsc_ = it->second;
    return;
  }
  // Create a new resource.
  // This will search a best algorithm given config.
  rsc_ = make_shared<CudnnConvResource>(desc);
  rsc.insert({desc, rsc_}); // Register the created resource to global.
}

template <class T>
void ConvolutionCudaCudnn<T>::forward_impl(const Variables &inputs,
                                           const Variables &outputs) {
  cuda_set_device(std::stoi(this->ctx_.device_id));
  const Tw *x = inputs[0]->get_data_pointer<Tw>(this->ctx_);
  const Tw *w = inputs[1]->get_data_pointer<Tw>(this->ctx_);
  Tw *y = outputs[0]->cast_data_and_get_pointer<Tw>(this->ctx_, true);
  auto alpha = get_cudnn_scalar_arg<T>(1);
  auto beta = get_cudnn_scalar_arg<T>(0);
  const Tw *b;
  if (inputs.size() == 3) {
    b = inputs[2]->get_data_pointer<Tw>(this->ctx_);
  }
  auto workspace_size = rsc_->workspace_size();
  unique_ptr<CudaCachedArray> workspace_arr;
  void *workspace{nullptr};
  if (workspace_size) {
    workspace_arr.reset(
        new CudaCachedArray(workspace_size, dtypes::BYTE, this->ctx_));
    workspace = workspace_arr->pointer<void>();
  }
#if CUDNN_VERSION >= 7000
  NBLA_CUDNN_CHECK(hipdnnConvolutionForward(
      cudnn_handle_, &alpha, rsc_->x_desc, x, rsc_->w_desc, w,
      rsc_->conv_desc.desc, rsc_->fwd_algo, workspace, rsc_->fwd_workspace_size,
      &beta, rsc_->y_desc, y));
  if (inputs.size() == 3) {
    NBLA_CUDNN_CHECK(hipdnnAddTensor(cudnn_handle_, &alpha, rsc_->b_desc, b,
                                    &alpha, rsc_->y_desc, y));
  }
#else
  for (int g = 0; g < this->group_; ++g) {
    NBLA_CUDNN_CHECK(hipdnnConvolutionForward(
        cudnn_handle_, &alpha, rsc_->x_desc, x + x_offset_ * g, rsc_->w_desc,
        w + w_offset_ * g, rsc_->conv_desc.desc, rsc_->fwd_algo, workspace,
        rsc_->fwd_workspace_size, &beta, rsc_->y_desc, y + y_offset_ * g));
    if (inputs.size() == 3) {
      // TODO: Bias addition should be outside of the loop. In that case,
      // b_desc and y_desc must be whole image descriptor.
      NBLA_CUDNN_CHECK(hipdnnAddTensor(cudnn_handle_, &alpha, rsc_->b_desc,
                                      b + b_offset_ * g, &alpha, rsc_->y_desc,
                                      y + y_offset_ * g));
    }
  }
#endif
}

template <class T>
void ConvolutionCudaCudnn<T>::backward_impl(const Variables &inputs,
                                            const Variables &outputs,
                                            const vector<bool> &propagate_down,
                                            const vector<bool> &accum) {
  if (!(propagate_down[0] || propagate_down[1] ||
        (inputs.size() == 3 && propagate_down[2]))) {
    return;
  }
  cuda_set_device(std::stoi(this->ctx_.device_id));
  const T *dy = outputs[0]->get_grad_pointer<T>(this->ctx_);
  const T *x;
  const T *w;
  T *dx, *dw, *db;
  if (propagate_down[0]) {
    w = inputs[1]->get_data_pointer<T>(this->ctx_);
    dx = inputs[0]->cast_grad_and_get_pointer<T>(this->ctx_, !accum[0]);
  }
  if (propagate_down[1]) {
    x = inputs[0]->get_data_pointer<T>(this->ctx_);
    dw = inputs[1]->cast_grad_and_get_pointer<T>(this->ctx_, !accum[1]);
  }
  if (propagate_down[2]) {
    db = inputs[2]->cast_grad_and_get_pointer<T>(this->ctx_, !accum[2]);
  }
  auto alpha = get_cudnn_scalar_arg<T>(1);
  auto workspace_size = rsc_->workspace_size();
  unique_ptr<CudaCachedArray> workspace_arr, workspace_arr_dgrad;
  void *workspace{nullptr}, *workspace_dgrad{nullptr};
  if (workspace_size) {
    workspace_arr.reset(
        new CudaCachedArray(workspace_size, dtypes::BYTE, this->ctx_));
    workspace = workspace_arr->pointer<void>();
    workspace_arr_dgrad.reset(
        new CudaCachedArray(workspace_size, dtypes::BYTE, this->ctx_));
    workspace_dgrad = workspace_arr_dgrad->pointer<void>();
  }
#if CUDNN_VERSION >= 7000
  if (propagate_down[0]) {
    this->wait_default_on_dgrad();
    auto beta = get_cudnn_scalar_arg<T>(accum[0] ? 1 : 0);
    NBLA_CUDNN_CHECK(hipdnnConvolutionBackwardData(
        dgrad_handle_, &alpha, rsc_->w_desc, w, rsc_->y_desc, dy,
        rsc_->conv_dgrad_desc.desc, rsc_->bwd_data_algo, workspace_dgrad,
        rsc_->bwd_data_workspace_size, &beta, rsc_->x_desc, dx));
  }
  if (propagate_down[1]) {
    /** Note:
    * When the bwd of first layer convolution is slower, check the value of
    * beta.
    * In the case of beta = 1, Not first_layer_wgrad_kernel which is faster than
    * any others but a slower kernel would be called in cudnn API.
    */
    auto beta = get_cudnn_scalar_arg<T>(accum[1] ? 1 : 0);
    NBLA_CUDNN_CHECK(hipdnnConvolutionBackwardFilter(
        cudnn_handle_, &alpha, rsc_->x_desc, x, rsc_->y_desc, dy,
        rsc_->conv_wgrad_desc.desc, rsc_->bwd_filter_algo, workspace,
        rsc_->bwd_filter_workspace_size, &beta, rsc_->w_desc, dw));
  }
  if (inputs.size() == 3 && propagate_down[2]) {
    auto beta = get_cudnn_scalar_arg<T>(accum[2] ? 1 : 0);
    NBLA_CUDNN_CHECK(hipdnnConvolutionBackwardBias(
        cudnn_handle_, &alpha, rsc_->y_desc, dy, &beta, rsc_->b_desc, db));
  }
  this->wait_dgrad_on_default();
#else
  for (int g = 0; g < this->group_; ++g) {
    if (propagate_down[0]) {
      auto beta = get_cudnn_scalar_arg<T>(accum[0] ? 1 : 0);
      NBLA_CUDNN_CHECK(hipdnnConvolutionBackwardData(
          cudnn_handle_, &alpha, rsc_->w_desc, w + w_offset_ * g, rsc_->y_desc,
          dy + y_offset_ * g, rsc_->conv_dgrad_desc.desc, rsc_->bwd_data_algo,
          workspace, rsc_->bwd_data_workspace_size, &beta, rsc_->x_desc,
          dx + x_offset_ * g));
    }
    if (propagate_down[1]) {
      auto beta = get_cudnn_scalar_arg<T>(accum[1] ? 1 : 0);
      NBLA_CUDNN_CHECK(hipdnnConvolutionBackwardFilter(
          cudnn_handle_, &alpha, rsc_->x_desc, x + x_offset_ * g, rsc_->y_desc,
          dy + y_offset_ * g, rsc_->conv_wgrad_desc.desc, rsc_->bwd_filter_algo,
          workspace, rsc_->bwd_filter_workspace_size, &beta, rsc_->w_desc,
          dw + w_offset_ * g));
    }
    if (inputs.size() == 3 && propagate_down[2]) {
      auto beta = get_cudnn_scalar_arg<T>(accum[2] ? 1 : 0);
      NBLA_CUDNN_CHECK(hipdnnConvolutionBackwardBias(
          cudnn_handle_, &alpha, rsc_->y_desc, dy + y_offset_ * g, &beta,
          rsc_->b_desc, db + b_offset_ * g));
    }
  }
#endif
}

// Manually selecting algorithms is not supported for now.
/*
// Basically this functions is not invoked,
// because it is chosen by hipdnnGetConvolutionForwardAlgorithm()
template <class T>
void ConvolutionCudaCudnn<T>::set_cudnn_convolution_forward_algorithm(
    std::string algorithm) {
  if (algorithm == "ALGO_IMPLICIT_GEMM") {
    fwd_algo_ = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM;
  } else if (algorithm == "ALGO_IMPLICIT_PRECOMP_GEMM") {
    fwd_algo_ = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;
  } else if (algorithm == "ALGO_GEMM") {
    fwd_algo_ = HIPDNN_CONVOLUTION_FWD_ALGO_GEMM;
  } else if (algorithm == "ALGO_DIRECT") {
    fwd_algo_ = HIPDNN_CONVOLUTION_FWD_ALGO_DIRECT;
  } else if (algorithm == "ALGO_FFT") {
    fwd_algo_ = HIPDNN_CONVOLUTION_FWD_ALGO_FFT;
  } else if (algorithm == "ALGO_FFT_TILING") {
    fwd_algo_ = HIPDNN_CONVOLUTION_FWD_ALGO_FFT_TILING;
  }
#if CUDNN_VERSION >= 5000
  else if (algorithm == "ALGO_WINOGRAD") {
    fwd_algo_ = HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD;
  }
#endif
  else {
    NBLA_ERROR(error_code::target_specific,
               "Specified unsupported forward algorithm");
  }
}

// Basically this functions is not invoked,
// because it is chosen by hipdnnGetConvolutionBackwardFilterAlgorithm()
template <class T>
void ConvolutionCudaCudnn<T>::set_cudnn_convolution_backward_filter_algorithm(
    std::string algorithm) {
  if (algorithm == "ALGO_0") {
    bwd_filter_algo_ = HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_0;
  } else if (algorithm == "ALGO_1") {
    bwd_filter_algo_ = HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_1;
  } else if (algorithm == "ALGO_FFT") {
    bwd_filter_algo_ = HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_FFT;
  } else if (algorithm == "ALGO_3") {
    bwd_filter_algo_ = HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_3;
  }
#if CUDNN_VERSION >= 5100
  else if (algorithm == "ALGO_WINOGRAD_NONFUSED") {
    bwd_filter_algo_ = HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_WINOGRAD_NONFUSED;
  }
#endif
  else {
    NBLA_ERROR(error_code::target_specific,
               "Specified unsupported backward filter algorithm");
  }
}

// Basically this functions is not invoked,
// because it is chosen by hipdnnGetConvolutionBackwardDataAlgorithm()
template <class T>
void ConvolutionCudaCudnn<T>::set_cudnn_convolution_backward_data_algorithm(
    std::string algorithm) {
  if (algorithm == "ALGO_0") {
    bwd_data_algo_ = HIPDNN_CONVOLUTION_BWD_DATA_ALGO_0;
  } else if (algorithm == "ALGO_1") {
    bwd_data_algo_ = HIPDNN_CONVOLUTION_BWD_DATA_ALGO_1;
  } else if (algorithm == "ALGO_FFT") {
    bwd_data_algo_ = HIPDNN_CONVOLUTION_BWD_DATA_ALGO_FFT;
  } else if (algorithm == "ALGO_FFT_TILING") {
    bwd_data_algo_ = HIPDNN_CONVOLUTION_BWD_DATA_ALGO_FFT_TILING;
  }
#if CUDNN_VERSION >= 5000
  else if (algorithm == "ALGO_WINOGRAD") {
    bwd_data_algo_ = HIPDNN_CONVOLUTION_BWD_DATA_ALGO_WINOGRAD;
  }
#if CUDNN_VERSION >= 5100
  else if (algorithm == "ALGO_WINOGRAD_NONFUSED") {
    bwd_data_algo_ = HIPDNN_CONVOLUTION_BWD_DATA_ALGO_WINOGRAD_NONFUSED;
  }
#endif
#endif
  else {
    NBLA_ERROR(error_code::target_specific,
               "Specified unsupported backward data algorithm");
  }
}

template <class T>
void ConvolutionCudaCudnn<T>::set_cudnn_convolution_mode(std::string mode) {
  if (mode == "CONVOLUTION") {
    mode_ = HIPDNN_CONVOLUTION;
  } else if (mode == "CROSS_CORRELATION") {
    mode_ = HIPDNN_CROSS_CORRELATION;
  } else {
    NBLA_ERROR(error_code::target_specific, "Specified unsupported algorithm");
  }
}
*/
}
