// Copyright (c) 2017 Sony Corporation. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <nbla/array.hpp>
#include <nbla/cuda/array/cuda_array.hpp>
#include <nbla/cuda/common.hpp>
#include <nbla/cuda/cudnn/cudnn.hpp>
#include <nbla/cuda/cudnn/function/prod.hpp>
#include <nbla/variable.hpp>

#if CUDNN_VERSION >= 6000
namespace nbla {

template <typename T>
void ProdCudaCudnn<T>::setup_impl(const Variables &inputs,
                                  const Variables &outputs) {
  ProdCuda<T>::setup_impl(inputs, outputs);
  cuda_set_device(this->device_);

  NBLA_CUDNN_CHECK(hipdnnSetReduceTensorDescriptor(
      this->reduce_desc_, HIPDNN_REDUCE_TENSOR_MUL, HIPDNN_DATA_FLOAT,
      HIPDNN_PROPAGATE_NAN, HIPDNN_REDUCE_TENSOR_NO_INDICES, HIPDNN_8BIT_INDICES));

  std::vector<int> x_shape, y_shape;
  x_shape.reserve(CUDNN_DIM_MAX);
  y_shape.reserve(CUDNN_DIM_MAX);

  for (auto dim : inputs[0]->shape()) {
    x_shape.push_back(static_cast<int>(dim));
    y_shape.push_back(static_cast<int>(dim));
  }
  for (auto axis : this->axes_)
    y_shape.at(axis) = 1;

  this->same_in_out_shape_ = (y_shape == x_shape) ? true : false;

  if (!this->same_in_out_shape_) {
    cudnn_set_tensor_descriptor<T>(this->x_desc_, x_shape);
    cudnn_set_tensor_descriptor<T>(this->y_desc_, y_shape);

    auto cudnn_handle_manager = SingletonManager::get<CudnnHandleManager>();
    auto cudnn_handle = cudnn_handle_manager->handle(this->device_);
    NBLA_CUDNN_CHECK(hipdnnGetReductionWorkspaceSize(
        cudnn_handle, this->reduce_desc_, this->x_desc_, this->y_desc_,
        &this->workspace_size_));
  }
}

template <typename T>
void ProdCudaCudnn<T>::forward_impl(const Variables &inputs,
                                    const Variables &outputs) {
  if ((!this->f_transpose_) || (inputs[0]->shape().size() > CUDNN_DIM_MAX)) {
    ProdCuda<T>::forward_impl(inputs, outputs);
    return;
  }
  if (this->same_in_out_shape_) {
    const Array *x = inputs[0]->data()->get(get_dtype<Tcu>(), this->ctx_);
    Array *y = outputs[0]->data()->cast(get_dtype<Tcu>(), this->ctx_, true);
    y->copy_from(x);
    return;
  }
  cuda_set_device(this->device_);
  auto cudnn_handle_manager = SingletonManager::get<CudnnHandleManager>();
  auto cudnn_handle = cudnn_handle_manager->handle(this->device_);

  unique_ptr<CudaCachedArray> workspace_arr;
  void *workspace{nullptr};
  if (this->workspace_size_) {
    workspace_arr.reset(
        new CudaCachedArray(this->workspace_size_, dtypes::BYTE, this->ctx_));
    workspace = workspace_arr->pointer<void>();
  }

  auto x_data = inputs[0]->get_data_pointer<Tcu>(this->ctx_);
  auto y_data = outputs[0]->cast_data_and_get_pointer<Tcu>(this->ctx_, true);
  float alpha = 1.0f, beta = 0.0f;

  NBLA_CUDNN_CHECK(hipdnnReduceTensor(cudnn_handle, this->reduce_desc_, nullptr,
                                     0UL, workspace, this->workspace_size_,
                                     &alpha, this->x_desc_, x_data, &beta,
                                     this->y_desc_, y_data));
}

template <typename T>
void ProdCudaCudnn<T>::backward_impl(const Variables &inputs,
                                     const Variables &outputs,
                                     const vector<bool> &propagate_down,
                                     const vector<bool> &accum) {
  if (propagate_down[0] && this->f_transpose_) {
    this->f_transpose_->forward(inputs, Variables{this->o_transpose_.get()});
  }
  ProdCuda<T>::backward_impl(inputs, outputs, propagate_down, accum);
}

} // namespace nbla
#endif // CUDNN_VERSION >= 6000
