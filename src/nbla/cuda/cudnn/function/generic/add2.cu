// Copyright (c) 2017 Sony Corporation. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

/** Add2
 */

#include <nbla/array.hpp>
#include <nbla/cuda/common.hpp>
#include <nbla/cuda/cudnn/cudnn.hpp>
#include <nbla/cuda/cudnn/function/add2.hpp>
#include <nbla/cuda/function/bc_add2.hpp>
#include <nbla/variable.hpp>

#include <algorithm>
#include <memory>

namespace nbla {

template <typename T>
void Add2CudaCudnn<T>::setup_impl(const Variables &inputs,
                                  const Variables &outputs) {
  if (inputs[0]->shape() != inputs[1]->shape()) {
    // Trying to fallback to broadcastable Add2.
    this->fall_back_func_ = std::shared_ptr<Function>(
        new BcAdd2Cuda<T>(this->ctx_, this->inplace_));
    this->fall_back_func_->setup(inputs, outputs);
    return;
  }

  Add2<T>::setup_impl(inputs, outputs);
  cudnn_handle_ = SingletonManager::get<CudnnHandleManager>()->handle(device_);
  NBLA_CUDNN_CHECK(hipdnnSetTensor4dDescriptor(input_desc_, HIPDNN_TENSOR_NCHW,
                                              cudnn_data_type<T>::type(), 1, 1,
                                              1, inputs[0]->size()));
  NBLA_CUDNN_CHECK(hipdnnSetTensor4dDescriptor(output_desc_, HIPDNN_TENSOR_NCHW,
                                              cudnn_data_type<T>::type(), 1, 1,
                                              1, outputs[0]->size()));
}

template <typename T>
void Add2CudaCudnn<T>::forward_impl(const Variables &inputs,
                                    const Variables &outputs) {
  cuda_set_device(std::stoi(this->ctx_.device_id));
  const Tw *x0 = inputs[0]->get_data_pointer<Tw>(this->ctx_);
  const Tw *x1 = inputs[1]->get_data_pointer<Tw>(this->ctx_);
  Tw *y =
      outputs[0]->cast_data_and_get_pointer<Tw>(this->ctx_, !this->inplace_);
  auto alpha = get_cudnn_scalar_arg<T>(1);
  auto beta = get_cudnn_scalar_arg<T>(1);
  if (x0 == y) {
#if CUDNN_VERSION >= 4000
    NBLA_CUDNN_CHECK(hipdnnAddTensor(cudnn_handle_, &alpha, input_desc_, x1,
                                    &beta, output_desc_, y));
#else
    NBLA_CUDNN_CHECK(hipdnnAddTensor(cudnn_handle_, CUDNN_ADD_FULL_TENSOR,
                                    &alpha, input_desc_, x1, &beta,
                                    output_desc_, y));
#endif
  } else if (x1 == y) {
#if CUDNN_VERSION >= 4000
    NBLA_CUDNN_CHECK(hipdnnAddTensor(cudnn_handle_, &alpha, input_desc_, x0,
                                    &beta, output_desc_, y));
#else
    NBLA_CUDNN_CHECK(hipdnnAddTensor(cudnn_handle_, CUDNN_ADD_FULL_TENSOR,
                                    &alpha, input_desc_, x0, &beta,
                                    output_desc_, y));
#endif
  } else {
    Add2Cuda<T>::forward_impl(inputs, outputs);
  }
}

template <typename T>
void Add2CudaCudnn<T>::backward_impl(const Variables &inputs,
                                     const Variables &outputs,
                                     const vector<bool> &propagate_down,
                                     const vector<bool> &accum) {
  cuda_set_device(std::stoi(this->ctx_.device_id));
  Tw *dx0 = inputs[0]->cast_grad_and_get_pointer<Tw>(
      this->ctx_, !(this->inplace_ || accum[0]));
  Tw *dx1 = inputs[1]->cast_grad_and_get_pointer<Tw>(this->ctx_, !accum[1]);
  const Tw *dy = outputs[0]->get_grad_pointer<Tw>(this->ctx_);
  auto alpha = get_cudnn_scalar_arg<T>(1);

  if (dx0 != dy && propagate_down[0]) {
    auto beta = get_cudnn_scalar_arg<T>(accum[0] ? 1 : 0);
#if CUDNN_VERSION >= 4000
    NBLA_CUDNN_CHECK(hipdnnAddTensor(cudnn_handle_, &alpha, input_desc_, dy,
                                    &beta, output_desc_, dx0));
#else
    NBLA_CUDNN_CHECK(hipdnnAddTensor(cudnn_handle_, CUDNN_ADD_FULL_TENSOR,
                                    &alpha, input_desc_, dy, &beta,
                                    output_desc_, dx0));
#endif
  }
  // dx1 == dy never happens actually.
  if (dx1 != dy && propagate_down[1]) {
    auto beta = get_cudnn_scalar_arg<T>(accum[1] ? 1 : 0);
#if CUDNN_VERSION >= 4000
    NBLA_CUDNN_CHECK(hipdnnAddTensor(cudnn_handle_, &alpha, input_desc_, dy,
                                    &beta, output_desc_, dx1));
#else
    NBLA_CUDNN_CHECK(hipdnnAddTensor(cudnn_handle_, CUDNN_ADD_FULL_TENSOR,
                                    &alpha, input_desc_, dy, &beta,
                                    output_desc_, dx1));
#endif
  }
}
}
