// Copyright 2018,2019,2020,2021 Sony Corporation.
// Copyright 2021,2022 Sony Group Corporation.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

/** ReLU
 */

#include <algorithm>
#include <nbla/array.hpp>
#include <nbla/cuda/common.hpp>
#include <nbla/cuda/cudnn/cudnn.hpp>
#include <nbla/cuda/cudnn/function/relu.hpp>
#include <nbla/variable.hpp>

namespace nbla {

// This implementation can be activated when the relu in
// build-tools/code_generator/function_types.yaml is activated.
// See also include/nbla/cuda/cudnn/function/generic/relu.cu
//
// ReLUCudaCudnn requres inputs[0] (x) and outputs[0] (y), but ReLUCuda only
// requires y. Therefore ReLUCuda has an advantage of memory usage over
// ReLUCudaCudnn. That is why ReLUCudaCudnn was removed.
#if 0
template <typename T>
void ReLUCudaCudnn<T>::setup_impl(const Variables &inputs,
                                  const Variables &outputs) {
  outputs[0]->reshape(inputs[0]->shape(), true);
  cudnn_handle_ = SingletonManager::get<CudnnHandleManager>()->handle(device_);
  NBLA_CUDNN_CHECK(hipdnnSetTensor4dDescriptor(input_desc_, HIPDNN_TENSOR_NCHW,
                                              cudnn_data_type<T>::type(), 1, 1,
                                              1, inputs[0]->size()));
  NBLA_CUDNN_CHECK(hipdnnSetTensor4dDescriptor(output_desc_, HIPDNN_TENSOR_NCHW,
                                              cudnn_data_type<T>::type(), 1, 1,
                                              1, outputs[0]->size()));
}

template <typename T>
void ReLUCudaCudnn<T>::forward_impl(const Variables &inputs,
                                    const Variables &outputs) {
  cuda_set_device(std::stoi(this->ctx_.device_id));
  const Tw *x = inputs[0]->get_data_pointer<Tw>(this->ctx_);
  Tw *y = outputs[0]->cast_data_and_get_pointer<Tw>(this->ctx_, true);
  auto alpha = get_cudnn_scalar_arg<T>(1);
  auto beta = get_cudnn_scalar_arg<T>(0);
#if CUDNN_VERSION >= 5000
  NBLA_CUDNN_CHECK(hipdnnActivationForward(cudnn_handle_, activation_desc_,
                                          &alpha, input_desc_, x, &beta,
                                          output_desc_, y));
#else
  NBLA_CUDNN_CHECK(cudnnActivationForward_v4(cudnn_handle_, activation_desc_,
                                             &alpha, input_desc_, x, &beta,
                                             output_desc_, y));
#endif
}

template <typename T>
void ReLUCudaCudnn<T>::backward_impl(const Variables &inputs,
                                     const Variables &outputs,
                                     const vector<bool> &propagate_down,
                                     const vector<bool> &accum) {
  if (!propagate_down[0]) {
    return;
  }
  cuda_set_device(std::stoi(this->ctx_.device_id));
  const Tw *y = outputs[0]->get_data_pointer<Tw>(this->ctx_);
  Tw *dx = inputs[0]->cast_grad_and_get_pointer<Tw>(this->ctx_, !accum[0]);
  const Tw *dy = outputs[0]->get_grad_pointer<Tw>(this->ctx_);
  const Tw *x = inputs[0]->get_data_pointer<Tw>(this->ctx_);
  auto alpha = get_cudnn_scalar_arg<T>(1);
  auto beta = get_cudnn_scalar_arg<T>(accum[0] ? 1 : 0);
#if CUDNN_VERSION >= 5000
  NBLA_CUDNN_CHECK(hipdnnActivationBackward(
      cudnn_handle_, activation_desc_, &alpha, output_desc_, y,
      this->output_desc_, dy, input_desc_, x, &beta, input_desc_, dx));
#else
  NBLA_CUDNN_CHECK(cudnnActivationBackward_v4(
      cudnn_handle_, activation_desc_, &alpha, output_desc_, y,
      this->output_desc_, dy, input_desc_, x, &beta, input_desc_, dx));
#endif
}
#endif
}
