#include "hip/hip_runtime.h"
// Copyright (c) 2017 Sony Corporation. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <nbla/cuda/common.hpp>
#include <nbla/cuda/solver/adagrad.hpp>

#include "./clip_grad.cuh"
#include "./mixed_precision_training.cuh"
#include "./weight_decay.cuh"

namespace nbla {

template <typename T>
__global__ void kernel_adagrad_update(const int num, T *data, const T *grad,
                                      T *g, const float lr, const float eps) {
  NBLA_CUDA_KERNEL_LOOP(idx, num) {
    g[idx] += grad[idx] * grad[idx];
    data[idx] -= lr * grad[idx] / (sqrt(g[idx]) + eps);
  }
}

template <typename T>
void AdagradCuda<T>::update_impl(const string &key, VariablePtr param) {
  Size_t size = param->size();
  auto &state = this->states_.at(key);
  VariablePtr g_ = state.pstate["v"];
  T *g = g_->cast_data_and_get_pointer<T>(this->ctx_);
  auto &t = state.t;
  const T *grad = param->get_grad_pointer<T>(this->ctx_);
  T *data = param->cast_data_and_get_pointer<T>(this->ctx_);
  t = std::min(t + 1, std::numeric_limits<uint32_t>::max() - 1);
  NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_adagrad_update, size, data, grad, g,
                                 this->lr_, this->eps_);
}

NBLA_DEF_WEIGHT_DECAY(AdagradCuda, weight_decay_cuda);
NBLA_DEF_CLIP_GRAD_BY_NORM(AdagradCuda, clip_grad_by_norm_cuda);
NBLA_DEF_CHECK_INF_GRAD(AdagradCuda, check_inf_grad_cuda);
NBLA_DEF_CHECK_NAN_GRAD(AdagradCuda, check_nan_grad_cuda);
NBLA_DEF_CHECK_INF_OR_NAN_GRAD(AdagradCuda, check_inf_or_nan_grad_cuda);
NBLA_DEF_SCALE_GRAD(AdagradCuda, scale_grad_impl_cuda);
}
