#include "hip/hip_runtime.h"
// Copyright (c) 2017 Sony Corporation. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <nbla/cuda/common.hpp>
#include <nbla/cuda/solver/amsbound.hpp>

#include "./clip_grad.cuh"
#include "./mixed_precision_training.cuh"
#include "./weight_decay.cuh"

namespace nbla {

template <typename T>
__global__ void kernel_amsbound_update(const int num, T *theta, T *m, T *v,
                                       T *v_hat, const T *g, float alpha_t,
                                       const float beta1, const float beta2,
                                       const float eps, const float final_lr,
                                       const float gamma) {
  NBLA_CUDA_KERNEL_LOOP(s, num) {
    // Updating running mean and var.
    m[s] = beta1 * m[s] + (1 - beta1) * g[s];
    v[s] = beta2 * v[s] + (1 - beta2) * g[s] * g[s];
    v_hat[s] = max(v_hat[s], v[s]);
    T lower_bound = final_lr * (1 - 1 / (gamma + 1));
    T upper_bound = final_lr * (1 + 1 / gamma);
    T denom = std::sqrt(v_hat[s]) + eps;
    T eta = min(upper_bound, max(alpha_t / denom, lower_bound));
    // Update parameters.
    theta[s] = theta[s] - eta * m[s];
  }
}

template <typename T>
void AMSBoundCuda<T>::update_impl(const string &key, VariablePtr param) {
  cuda_set_device(std::stoi(this->ctx_.device_id));
  Size_t size = param->size();
  auto &state = this->states_.at(key);
  auto &t = state.t;
  const T *g = param->get_grad_pointer<T>(this->ctx_);
  shared_ptr<Variable> mean_ = state.pstate["m"]; // To prevent compile error.
  shared_ptr<Variable> var_ = state.pstate["v"];  // To prevent compile error.
  shared_ptr<Variable> var_hat_ =
      state.pstate["v_hat"]; // To prevent compile error.
  T *m = mean_->cast_data_and_get_pointer<T>(this->ctx_);
  T *v = var_->cast_data_and_get_pointer<T>(this->ctx_);
  T *v_hat = var_hat_->cast_data_and_get_pointer<T>(this->ctx_);
  T *theta = param->cast_data_and_get_pointer<T>(this->ctx_);
  t = std::min(t + 1, std::numeric_limits<uint32_t>::max() - 1);
  const T bias_correction = std::sqrt(1 - std::pow(this->beta2_, t)) /
                            (1 - std::pow(this->beta1_, t));
  T alpha_t = this->alpha_ * (this->bias_correction_ ? bias_correction : 1);
  T final_lr = this->final_lr_ * (this->alpha_ / this->init_alpha_);
  NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_amsbound_update, size, theta, m, v,
                                 v_hat, g, alpha_t, this->beta1_, this->beta2_,
                                 this->eps_, final_lr, this->gamma_);
}
NBLA_DEF_WEIGHT_DECAY(AMSBoundCuda, weight_decay_cuda);
NBLA_DEF_CLIP_GRAD_BY_NORM(AMSBoundCuda, clip_grad_by_norm_cuda);
NBLA_DEF_CHECK_INF_GRAD(AMSBoundCuda, check_inf_grad_cuda);
NBLA_DEF_CHECK_NAN_GRAD(AMSBoundCuda, check_nan_grad_cuda);
NBLA_DEF_CHECK_INF_OR_NAN_GRAD(AMSBoundCuda, check_inf_or_nan_grad_cuda);
NBLA_DEF_SCALE_GRAD(AMSBoundCuda, scale_grad_impl_cuda);
}
