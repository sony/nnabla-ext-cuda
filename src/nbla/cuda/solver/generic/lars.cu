#include "hip/hip_runtime.h"
// Copyright (c) 2017 Sony Corporation. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <cassert>
#include <queue>

#include <nbla/cuda/array/cuda_array.hpp>
#include <nbla/cuda/common.hpp>
#include <nbla/cuda/cublas.hpp>
#include <nbla/cuda/cuda.hpp>
#include <nbla/cuda/solver/lars.hpp>
#include <nbla/cuda/utils/block_reduce.cuh>

#include "./clip_grad.cuh"
#include "./mixed_precision_training.cuh"
#include "./weight_decay.cuh"

namespace nbla {

constexpr int blocks = 1024; /* max blocks */

template <typename T>
__global__ void kernel_reduce_pow2_per_block(const int N, const T *x1, T *buff1,
                                             const T *x2, T *buff2) {
  typedef typename CudaTypeForceFloat<T>::type AccT;
  AccT thread_data1 = 0;
  NBLA_CUDA_KERNEL_LOOP(i, N) { thread_data1 += (AccT)x1[i] * (AccT)x1[i]; }
  thread_data1 = blockReduceSum(thread_data1);
  if (threadIdx.x == 0) {
    buff1[blockIdx.x] = thread_data1;
  }

  AccT thread_data2 = 0;
  NBLA_CUDA_KERNEL_LOOP(i, N) { thread_data2 += (AccT)x2[i] * (AccT)x2[i]; }
  thread_data2 = blockReduceSum(thread_data2);
  if (threadIdx.x == 0) {
    buff2[blockIdx.x] = thread_data2;
  }
}
template <typename T>
__global__ void kernel_reduce_per_block(const int N, const T *x1, T *buff1,
                                        const T *x2, T *buff2) {
  typedef typename CudaTypeForceFloat<T>::type AccT;
  AccT thread_data1 = 0;
  NBLA_CUDA_KERNEL_LOOP(i, N) { thread_data1 += (AccT)x1[i]; }
  thread_data1 = blockReduceSum(thread_data1);
  if (threadIdx.x == 0) {
    buff1[blockIdx.x] = thread_data1;
  }

  AccT thread_data2 = 0;
  NBLA_CUDA_KERNEL_LOOP(i, N) { thread_data2 += (AccT)x2[i]; }
  thread_data2 = blockReduceSum(thread_data2);
  if (threadIdx.x == 0) {
    buff2[blockIdx.x] = thread_data2;
  }
}
template <typename T>
void sq_sum(hipStream_t stream, const int num, const T *data, T *buff1,
            T *sq_data, const T *grad, T *buff2, T *sq_grad) {
  if (num >= 1024) {
    int blocks = min(NBLA_CUDA_GET_BLOCKS(num), /*max blocks*/ 1024);
    kernel_reduce_pow2_per_block<<<blocks, NBLA_CUDA_NUM_THREADS, 0, stream>>>(
        num, data, buff1, grad, buff2);
    kernel_reduce_per_block<<<1, 1024, 0, stream>>>(blocks, buff1, sq_data,
                                                    buff2, sq_grad);
  } else {
    kernel_reduce_pow2_per_block<<<1, 1024, 0, stream>>>(num, data, sq_data,
                                                         grad, sq_grad);
  }
}

template <typename T>
__global__ void kernel_lars_update(const int num, T *data, const T *grad, T *v,
                                   T *d_sq, T *g_sq, float lr, float momentum,
                                   float decay_rate, float coefficient,
                                   float eps) {
  /* Calculate L2 norm */
  auto g_norm = std::sqrt(*g_sq);
  auto d_norm = std::sqrt(*d_sq);

  /* Calculate local learning rate */
  auto x = g_norm + decay_rate * d_norm;
  if (x < eps) {
    x += eps;
  }
  float local_lr = 1;
  if (d_norm >= eps) {
    local_lr = coefficient * d_norm / x;
  }

  // Update weight and momentum
  NBLA_CUDA_KERNEL_LOOP(idx, num) {
    v[idx] = momentum * v[idx] +
             lr * local_lr * (grad[idx] + decay_rate * data[idx]);
    data[idx] -= v[idx];
  }
}

template <typename T>
void LarsCuda<T>::update_impl(const string &key, VariablePtr param) {
  cuda_set_device(std::stoi(this->ctx_.device_id));

  typedef typename CudaType<T>::type Tc;
  dtypes dtype = get_dtype<Tc>();
  auto g_sq_arr = make_shared<NdArray>(Shape_t{1});
  auto d_sq_arr = make_shared<NdArray>(Shape_t{1});
  Tc *g_sq = g_sq_arr->cast(dtype, this->ctx_)->pointer<Tc>();
  Tc *d_sq = d_sq_arr->cast(dtype, this->ctx_)->pointer<Tc>();

  shared_ptr<CudaCachedArray> d_buff_arr =
      make_shared<CudaCachedArray>(blocks, dtype, this->ctx_);
  Tc *d_buff = d_buff_arr->pointer<Tc>();
  shared_ptr<CudaCachedArray> g_buff_arr =
      make_shared<CudaCachedArray>(blocks, dtype, this->ctx_);
  Tc *g_buff = g_buff_arr->pointer<Tc>();

  Size_t size = param->size();
  VariablePtr v_var = this->states_.at(key).pstate["v"];
  Tc *v = v_var->cast_data_and_get_pointer<Tc>(this->ctx_);
  Tc *data = param->cast_data_and_get_pointer<Tc>(this->ctx_);
  const Tc *grad = param->get_grad_pointer<Tc>(this->ctx_);

  /* calculate squared sum */
  sq_sum(nullptr, size, data, d_buff, d_sq, grad, g_buff, g_sq);

  NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(
      kernel_lars_update, size, data, grad, v, d_sq, g_sq, this->lr_,
      this->momentum_, this->decay_rate_, this->coefficient_, this->eps_);

  auto &t = this->states_.at(key).t;
  t = std::min(t + 1, std::numeric_limits<uint32_t>::max() - 1);
}

NBLA_DEF_CLIP_GRAD_BY_NORM(LarsCuda, clip_grad_by_norm_cuda);
NBLA_DEF_CHECK_INF_GRAD(LarsCuda, check_inf_grad_cuda);
NBLA_DEF_CHECK_NAN_GRAD(LarsCuda, check_nan_grad_cuda);
NBLA_DEF_CHECK_INF_OR_NAN_GRAD(LarsCuda, check_inf_or_nan_grad_cuda);
NBLA_DEF_SCALE_GRAD(LarsCuda, scale_grad_impl_cuda);
}
