#include "hip/hip_runtime.h"
// Copyright 2018,2019,2020,2021 Sony Corporation.
// Copyright 2021,2022 Sony Group Corporation.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

// dropout.cu

#include <nbla/array.hpp>
#include <nbla/cuda/common.hpp>
#include <nbla/cuda/function/dropout.hpp>
#include <nbla/variable.hpp>

namespace nbla {

template <typename T>
__global__ void kernel_dropout_forward(const int size, const float scale,
                                       const float p, const T *x, T *y,
                                       float *m) {
  NBLA_CUDA_KERNEL_LOOP(s, size) {
    m[s] = (m[s] > p) ? 1 : 0;
    y[s] = x[s] * m[s] * scale;
  }
}

template <typename T>
__global__ void kernel_dropout_recompute(const int size, const float scale,
                                         const float p, const T *x, T *y,
                                         const float *m) {
  NBLA_CUDA_KERNEL_LOOP(s, size) {
    // This operation is done when forward. m[s] = (m[s] > p) ? 1 : 0;
    y[s] = x[s] * m[s] * scale;
  }
}

template <typename T, bool accum>
__global__ void kernel_dropout_backward(const int size, const float scale,
                                        const T *dy, const float *m, T *dx) {
  NBLA_CUDA_KERNEL_LOOP(s, size) {
    dx[s] = (accum ? dx[s] : (T)0) + dy[s] * m[s] * scale;
  }
}

template <typename T>
void DropoutCuda<T>::setup_impl(const Variables &inputs,
                                const Variables &outputs) {
  Dropout<T>::setup_impl(inputs, outputs);
}

template <typename T>
void DropoutCuda<T>::setup_recompute_impl(const Variables &inputs,
                                          const Variables &outputs) {
  store_mask_for_recompute_ = true;
}

template <class T>
void DropoutCuda<T>::forward_impl(const Variables &inputs,
                                  const Variables &outputs) {
  cuda_set_device(std::stoi(this->ctx_.device_id));
  const Tc *x = inputs[0]->get_data_pointer<Tc>(this->ctx_);
  Tc *y = outputs[0]->cast_data_and_get_pointer<Tc>(this->ctx_, true);
  VariablePtr mask = this->mask_;
  float *m = mask->cast_data_and_get_pointer<float>(this->ctx_, true);
  hiprandGenerator_t &gen =
      this->seed_ == -1 ? SingletonManager::get<Cuda>()->curand_generator()
                        : curand_generator_;
  curand_generate_rand<float>(gen, 0.0f, 1.0f, m, inputs[0]->size());
  NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_dropout_forward, inputs[0]->size(),
                                 this->scale_, this->p_, x, y, m);
}

template <class T>
void DropoutCuda<T>::recompute_impl(const Variables &inputs,
                                    const Variables &outputs) {
  NBLA_CHECK(this->mask_->data()->array()->get_num_arrays(),
             error_code::unclassified,
             "The mask of Dropout must be stored in mask_ for recomputation. "
             "Please report this error to the NNabla developer team.");
  cuda_set_device(std::stoi(this->ctx_.device_id));
  const Tc *x = inputs[0]->get_data_pointer<Tc>(this->ctx_);
  Tc *y = outputs[0]->cast_data_and_get_pointer<Tc>(this->ctx_, true);
  VariablePtr mask = this->mask_;
  const float *m = mask->get_data_pointer<float>(this->ctx_);
  NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_dropout_recompute, inputs[0]->size(),
                                 this->scale_, this->p_, x, y, m);
}

template <class T>
void DropoutCuda<T>::backward_impl(const Variables &inputs,
                                   const Variables &outputs,
                                   const vector<bool> &propagate_down,
                                   const vector<bool> &accum) {
  if (!propagate_down[0]) {
    return;
  }
  cuda_set_device(std::stoi(this->ctx_.device_id));
  Tc *dx = inputs[0]->cast_grad_and_get_pointer<Tc>(this->ctx_, !accum[0]);
  const Tc *dy = outputs[0]->get_grad_pointer<Tc>(this->ctx_);
  VariablePtr mask = this->mask_;
  const float *m = mask->get_data_pointer<float>(this->ctx_);
  if (accum[0]) {
    NBLA_CUDA_LAUNCH_KERNEL_SIMPLE((kernel_dropout_backward<Tc, true>),
                                   inputs[0]->size(), this->scale_, dy, m, dx);
  } else {
    NBLA_CUDA_LAUNCH_KERNEL_SIMPLE((kernel_dropout_backward<Tc, false>),
                                   inputs[0]->size(), this->scale_, dy, m, dx);
  }

  this->clear_buffer();
}
}
