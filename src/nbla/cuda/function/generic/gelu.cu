#include "hip/hip_runtime.h"
// Copyright 2019,2020,2021 Sony Corporation.
// Copyright 2021,2022 Sony Group Corporation.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <nbla/cuda/array/cuda_array.hpp>
#include <nbla/cuda/common.hpp>
#include <nbla/cuda/function/gelu.hpp>
#include <nbla/cuda/function/utils/base_transform_unary.cuh>

#include <cmath>

#ifndef M_PI
#define M_PI (3.14159265358979323846)
#endif

namespace nbla {
NBLA_DEFINE_TRANSFORM_UNARY_CUDA(
    GELU, x / 2 * (1 + std::tanh((std::sqrt((T)(2 / M_PI)) *
                                  (x + (T)0.044715 * std::pow(x, (T)3))))),
    dy *(0.5 * (1 + std::tanh(std::sqrt((T)(2 / M_PI)) *
                              (x + (T)0.044715 * std::pow(x, T(3))))) +
         0.5 * x *
             (1 - std::pow(std::tanh(std::sqrt((T)(2 / M_PI)) *
                                     (x + (T)0.044715 * std::pow(x, T(3)))),
                           T(2))) *
             std::sqrt((T)(2 / M_PI)) * (1 + 0.134145 * std::pow(x, T(2)))),
    false, true);
}
