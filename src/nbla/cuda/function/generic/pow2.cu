#include "hip/hip_runtime.h"
// Copyright (c) 2017 Sony Corporation. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

// pow2.cu

#include <nbla/cuda/function/pow2.hpp>
#include <nbla/cuda/function/utils/base_transform_binary.cuh>

#include <cmath>

namespace nbla {

NBLA_DEFINE_TRANSFORM_BINARY_CUDA(
    Pow2, std::pow(x0, x1),
    dy *x1 *std::pow(inplace ? std::pow(y, 1 / x1) : x0, x1 - (T)1),
    dy *std::log(inplace ? std::pow(y, 1 / x1) : x0) *
        std::pow(inplace ? std::pow(y, 1 / x1) : x0, x1));
}
