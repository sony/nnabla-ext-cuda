#include "hip/hip_runtime.h"
// Copyright (c) 2017 Sony Corporation. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <nbla/array.hpp>
#include <nbla/variable.hpp>

#include <nbla/cuda/common.hpp>
#include <nbla/cuda/function/batch_normalization.hpp>
#include <nbla/cuda/limits.hpp>

//#include <nbla/cuda/function/kernel/batch_normalization.cuh>
#include "kernel/batch_normalization.cu"

#define BATCH_NORMALIZATION_USE_PARALLEL_REDUCTION

namespace nbla {

template <typename T>
void BatchNormalizationCuda<T>::setup_impl(const Variables &inputs,
                                           const Variables &outputs) {
  BatchNormalization<T>::setup_impl(inputs, outputs);
  v_dmean_.reshape(Shape_t{this->size1_}, true);
  v_dvar_.reshape(Shape_t{this->size1_}, true);
#ifdef BATCH_NORMALIZATION_USE_PARALLEL_REDUCTION
  // setup for transpose
  const int ndim = inputs[0]->ndim();

  // for transpose
  v_axes_.reshape(Shape_t{ndim}, true);
  v_in_strides_.reshape(Shape_t{ndim}, true);
  v_out_strides_.reshape(Shape_t{ndim}, true);
  v_in_shape_.reshape(Shape_t{ndim}, true);
  v_out_shape_.reshape(Shape_t{ndim}, true);
  v_din_trans_.reshape(inputs[0]->shape(), true);

  // work memory for data of each axis
  v_inv_sqrt_variance_.reshape(Shape_t{this->size1_}, true);
  v_t_.reshape(Shape_t{this->size1_}, true);

  // work memory for each block data of shuffle reduction
  this->blocks =
      min((this->size02_ + NBLA_CUDA_NUM_THREADS - 1) / NBLA_CUDA_NUM_THREADS,
          1024);
  v_mean_reduction_space_.reshape(Shape_t{blocks}, true);
  v_variance_reduction_space_.reshape(Shape_t{blocks}, true);
  v_tmp_reduction_space_.reshape(Shape_t{blocks}, true);

  // make shape for transpose
  Context cpu; // CPU Context
  int *p_axes = v_axes_.cast_data_and_get_pointer<int>(cpu, true);
  int *p_in_strides = v_in_strides_.cast_data_and_get_pointer<int>(cpu, true);
  int *p_out_strides = v_out_strides_.cast_data_and_get_pointer<int>(cpu, true);
  int *p_out_shape = v_out_shape_.cast_data_and_get_pointer<int>(cpu, true);
  int *p_in_shape = v_in_shape_.cast_data_and_get_pointer<int>(cpu, true);
  for (int i = 0; i < ndim; p_axes[i] = i, ++i)
    ;
  if (this->axes_[0] != 0) {
    p_axes[0] = this->axes_[0];
    p_axes[this->axes_[0]] = 0;
  }
  Shape_t shape(ndim);
  for (int i = 0; i < ndim; ++i)
    shape[i] = inputs[0]->shape()[p_axes[i]];
  v_in_trans_.reshape(shape, true);
  for (int i = 0; i < ndim; ++i) {
    p_in_strides[i] = inputs[0]->strides()[i];
    p_out_strides[i] = v_in_trans_.strides()[i];
    p_in_shape[i] = inputs[0]->shape()[i];
    p_out_shape[i] = v_in_trans_.shape()[i];
  }
#endif
}

template <class T>
void BatchNormalizationCuda<T>::forward_impl(const Variables &inputs,
                                             const Variables &outputs) {
  cuda_set_device(std::stoi(this->ctx_.device_id));
  if (this->batch_stat_) { // Training mode.
    forward_impl_batch(inputs, outputs);
  } else { // Testing mode.
    forward_impl_global(inputs, outputs);
  }
}

template <class T>
void BatchNormalizationCuda<T>::forward_impl_batch(const Variables &inputs,
                                                   const Variables &outputs) {
  // Check whether it outputs batch mean and var.
  Variable *batch_mean = &this->mean_;
  Variable *batch_var = &this->var_;
  if (outputs.size() == 3) {
    batch_mean = outputs[1];
    batch_var = outputs[2];
  }
  // Inputs
  const Tc *x = inputs[0]->get_data_pointer<Tc>(this->ctx_);
  const Tc *beta = inputs[1]->get_data_pointer<Tc>(this->ctx_);
  const Tc *gamma = inputs[2]->get_data_pointer<Tc>(this->ctx_);
  // Output
  Tc *y = outputs[0]->cast_data_and_get_pointer<Tc>(this->ctx_, true);
  Tc *m =
      batch_mean->cast_data_and_get_pointer<Tc>(this->ctx_, true); // batch mean
  Tc *v =
      batch_var->cast_data_and_get_pointer<Tc>(this->ctx_, true); // batch varf
  // Inputs/Outputs
  Tc *rm = inputs[3]->cast_data_and_get_pointer<Tc>(this->ctx_); // running mean
  Tc *rv = inputs[4]->cast_data_and_get_pointer<Tc>(this->ctx_); // running var

#ifdef BATCH_NORMALIZATION_USE_PARALLEL_REDUCTION
  const int ndim = inputs[0]->ndim();
  auto get_ = [this](Variable &var) {
    return var.get_data_pointer<int>(this->ctx_);
  };
  auto get_data_ptr_ = [this](Variable &var) {
    return var.cast_data_and_get_pointer<Tc>(this->ctx_);
  };
  const int *axes = get_(this->v_axes_);
  const int *in_strides = get_(this->v_in_strides_);
  const int *out_strides = get_(this->v_out_strides_);
  const int *in_shape = get_(this->v_in_shape_);
  const int *out_shape = get_(this->v_out_shape_);
  Tc *in_trans = get_data_ptr_(this->v_in_trans_);
  Tc *mean_reduction_space = get_data_ptr_(this->v_mean_reduction_space_);
  Tc *variance_reduction_space =
      get_data_ptr_(this->v_variance_reduction_space_);
  Tc *inv_sqrt_variance = get_data_ptr_(this->v_inv_sqrt_variance_);
  forward_batch_parallel_reduction(
      this->size0_, this->size1_, this->size2_, ndim, axes, in_strides,
      in_shape, out_strides, out_shape, this->decay_rate_, this->eps_, x, gamma,
      beta, in_trans, m, v, rm, rv, y, mean_reduction_space,
      variance_reduction_space, inv_sqrt_variance);
#else
  forward_batch(this->size0_, this->size1_, this->size2_, this->decay_rate_,
                this->eps_, x, gamma, beta, m, v, rm, rv, y);
#endif
}

template <class T>
void BatchNormalizationCuda<T>::forward_impl_global(const Variables &inputs,
                                                    const Variables &outputs) {
  // Inputs
  const Tc *x = inputs[0]->get_data_pointer<Tc>(this->ctx_);
  const Tc *beta = inputs[1]->get_data_pointer<Tc>(this->ctx_);
  const Tc *gamma = inputs[2]->get_data_pointer<Tc>(this->ctx_);
  const Tc *rm = inputs[3]->get_data_pointer<Tc>(this->ctx_); // running mean
  const Tc *rv = inputs[4]->get_data_pointer<Tc>(this->ctx_); // running var
  // Output
  Tc *y = outputs[0]->cast_data_and_get_pointer<Tc>(this->ctx_, true);

  NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(
      forward_global_kernel, this->size1_ * this->size02_, this->size0_,
      this->size1_, this->size2_, this->size02_, this->size12_,
      this->decay_rate_, this->eps_, x, rm, rv, gamma, beta, y);
}

template <class T>
void BatchNormalizationCuda<T>::backward_impl(
    const Variables &inputs, const Variables &outputs,
    const vector<bool> &propagate_down, const vector<bool> &accum) {

  cuda_set_device(std::stoi(this->ctx_.device_id));
  if (this->batch_stat_) { // Training mode.
    backward_impl_batch(inputs, outputs, propagate_down, accum);
  } else { // Testing mode.
    this->backward_impl_global(inputs, outputs, propagate_down, accum);
  }
}

template <class T>
void BatchNormalizationCuda<T>::backward_impl_batch(
    const Variables &inputs, const Variables &outputs,
    const vector<bool> &propagate_down, const vector<bool> &accum) {
  if (!(propagate_down[0] || propagate_down[1] || propagate_down[2])) {
    return;
  }
  // Check whether it outputs batch mean/var.
  Variable *batch_mean = &this->mean_;
  Variable *batch_var = &this->var_;
  if (outputs.size() == 3) {
    batch_mean = outputs[1];
    batch_var = outputs[2];
  }
  // Common inputs wrt. gradient.
  const Tc *dy = outputs[0]->get_grad_pointer<Tc>(this->ctx_);
  const Tc *m = batch_mean->get_data_pointer<Tc>(this->ctx_);
  const Tc *v = batch_var->get_data_pointer<Tc>(this->ctx_);
  const Tc *x = inputs[0]->get_data_pointer<Tc>(this->ctx_);
  auto get_data_ptr_ = [this](Variable &var) {
    return var.cast_data_and_get_pointer<Tc>(this->ctx_);
  };
#ifdef BATCH_NORMALIZATION_USE_PARALLEL_REDUCTION
  int ndim = inputs[0]->ndim();
  auto get_ = [this](Variable &var) {
    return var.get_data_pointer<int>(this->ctx_);
  };
  const int *axes = get_(this->v_axes_);
  const int *in_strides = get_(this->v_in_strides_);
  const int *out_strides = get_(this->v_out_strides_);
  const int *in_shape = get_(this->v_in_shape_);
  const int *out_shape = get_(this->v_out_shape_);
  // TODO: write_only flags
  Tc *d_x_trans = get_data_ptr_(this->v_in_trans_);
  Tc *d_dy_trans = get_data_ptr_(this->v_din_trans_);
  Tc *mean_reduction_space = get_data_ptr_(this->v_mean_reduction_space_);
  Tc *variance_reduction_space =
      get_data_ptr_(this->v_variance_reduction_space_);
  Tc *inv_sqrt_variance = get_data_ptr_(this->v_inv_sqrt_variance_);
  NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(
      transpose_2value_kernel, this->size1_ * this->size02_, ndim, axes,
      in_strides, out_strides, out_shape, x, dy, d_x_trans, d_dy_trans);
#endif
  if (propagate_down[0]) {
    if (!accum[0])
      inputs[0]->grad()->zero(); // TODO: optimize this out if possible
    Tc *dx = inputs[0]->cast_grad_and_get_pointer<Tc>(this->ctx_, false);
    const Tc *g = inputs[2]->get_data_pointer<Tc>(this->ctx_);
    const Tc *dm = nullptr;
    const Tc *dv = nullptr;
    if (outputs.size() == 3) {
      dm = batch_mean->get_grad_pointer<Tc>(this->ctx_);
      dv = batch_var->get_grad_pointer<Tc>(this->ctx_);
    }
    Tc *dmean = get_data_ptr_(this->v_dmean_);
    Tc *dvar = get_data_ptr_(this->v_dvar_);
#ifdef BATCH_NORMALIZATION_USE_PARALLEL_REDUCTION
    Tc *tmp_reduction_space = get_data_ptr_(this->v_tmp_reduction_space_);
    Tc *t = get_data_ptr_(this->v_t_);
    backward_batch_data_parallel_reduction(
        this->size0_, this->size1_, this->size2_, ndim, axes, in_strides,
        in_shape, out_strides, out_shape, this->decay_rate_, this->eps_, dy, m,
        v, x, g, dm, dv, dx, mean_reduction_space, variance_reduction_space,
        tmp_reduction_space, dmean, dvar, t, inv_sqrt_variance, d_x_trans,
        d_dy_trans);
#else
    backward_batch_data(this->size0_, this->size1_, this->size2_,
                        this->decay_rate_, this->eps_, dy, m, v, x, g, dm, dv,
                        dx, dmean, dvar);
#endif
  }
  if (propagate_down[1] || propagate_down[2]) { // beta and gamma
    NBLA_CHECK(propagate_down[1] && propagate_down[2], error_code::value,
               "'need_grad' of beta and gamma must be the same.");
    if (!accum[1])
      inputs[1]->grad()->zero(); // TODO: optimize this out if possible
    if (!accum[2])
      inputs[2]->grad()->zero(); // TODO: optimize this out if possible
    Tc *db = inputs[1]->cast_grad_and_get_pointer<Tc>(this->ctx_, false);
    Tc *dg = inputs[2]->cast_grad_and_get_pointer<Tc>(this->ctx_, false);
#ifdef BATCH_NORMALIZATION_USE_PARALLEL_REDUCTION
    backward_batch_gamma_beta_parallel_reduction(
        this->size0_, this->size1_, this->size2_, d_dy_trans, m, v, d_x_trans,
        this->eps_, db, dg, mean_reduction_space, variance_reduction_space,
        inv_sqrt_variance);
#else
    NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(
        backward_batch_gamma_beta_kernel, this->size1_, this->size2_,
        this->size02_, this->size12_, this->eps_, dy, m, v, x, db, dg);
#endif
  }
}
}
