#include "hip/hip_runtime.h"
// Copyright (c) 2017 Sony Corporation. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <nbla/cuda/array/cuda_array.hpp>
#include <nbla/cuda/function/mean.hpp>
#include <nbla/cuda/math.hpp>
#include <nbla/cuda/utils/block_reduce.cuh>

namespace nbla {

template <typename T> using Accum = typename CudaTypeForceFloat<T>::type;

template <typename T>
__global__ void kernel_reduce_per_block(const int N, const T *x, T *buff,
                                        const Accum<T> scale = Accum<T>(1)) {
  Accum<T> thread_data = 0;
  NBLA_CUDA_KERNEL_LOOP(i, N) { thread_data += (Accum<T>)x[i]; }
  thread_data = blockReduceSum(thread_data);
  if (threadIdx.x == 0) {
    buff[blockIdx.x] = thread_data * scale;
  }
}

template <typename T>
void MeanCuda<T>::forward_impl_reduce(const T *x_, T *y_, int outer_size,
                                      int reduction_size) {
  const Tc *x = reinterpret_cast<const Tc *>(x_);
  Tc *y = reinterpret_cast<Tc *>(y_);
  cuda_set_device(this->device_);
  const Accum<T> scale = Accum<T>(1) / reduction_size;

  if (reduction_size / outer_size < 2048) {
    const Tc *ones =
        static_cast<const Tc *>(SingletonManager::get<NNabla>()->ones(
            reduction_size, get_dtype<Tc>(), this->ctx_));
    cuda_gemv<Tc>(this->device_, y, x, reduction_size, outer_size, true, ones,
                  reduction_size, scale, 0);
  } else if (reduction_size > 1024) {
    const int threads = NBLA_CUDA_NUM_THREADS;
    const int blocks = min(NBLA_CUDA_GET_BLOCKS(reduction_size), 1024);
    shared_ptr<CudaCachedArray> arr_buff =
        make_shared<CudaCachedArray>(blocks, get_dtype<Tc>(), this->ctx_);
    Tc *buff = arr_buff->pointer<Tc>();
    while (outer_size--) {
      kernel_reduce_per_block<Tc><<<blocks, threads>>>(reduction_size, x, buff,
                                                       scale);
      NBLA_CUDA_KERNEL_CHECK();
      kernel_reduce_per_block<Tc><<<1, 1024>>>(blocks, buff, y);
      NBLA_CUDA_KERNEL_CHECK();
      x += reduction_size;
      y += 1;
    }
  } else {
    while (outer_size--) {
      kernel_reduce_per_block<Tc><<<1, 1024>>>(reduction_size, x, y, scale);
      NBLA_CUDA_KERNEL_CHECK();
      x += reduction_size;
      y += 1;
    }
  }
}

template <typename T, bool accum>
__global__ void kernel_reduce_mean_backward(const int num, T *dx, const T *dy,
                                            T scale) {
  NBLA_CUDA_KERNEL_LOOP(idx, num) {
    dx[idx] = (accum ? dx[idx] : (T)0) + scale * (*dy);
  }
}

template <typename T>
void MeanCuda<T>::backward_impl_reduce(const T *dy_, T *dx_, int outer_size,
                                       int reduction_size, bool accum) {
  const Tc *dy = reinterpret_cast<const Tc *>(dy_);
  Tc *dx = reinterpret_cast<Tc *>(dx_);
  cuda_set_device(this->device_);
  if (outer_size == 1) {
    if (accum) {
      NBLA_CUDA_LAUNCH_KERNEL_SIMPLE((kernel_reduce_mean_backward<Tc, true>),
                                     reduction_size, dx, dy,
                                     (T)(1. / reduction_size));
    } else {
      NBLA_CUDA_LAUNCH_KERNEL_SIMPLE((kernel_reduce_mean_backward<Tc, false>),
                                     reduction_size, dx, dy,
                                     (T)(1. / reduction_size));
    }
    return;
  }
  const Tc *ones =
      static_cast<const Tc *>(SingletonManager::get<NNabla>()->ones(
          reduction_size, get_dtype<Tc>(), this->ctx_));
  cuda_gemm<Tc>(this->device_, dx, true, dy, outer_size, 1, false, ones, 1,
                reduction_size, false, 1. / reduction_size, accum ? 1 : 0);
}
}
