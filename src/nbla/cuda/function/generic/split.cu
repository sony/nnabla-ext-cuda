#include "hip/hip_runtime.h"
// Copyright 2018,2019,2020,2021 Sony Corporation.
// Copyright 2022 Sony Group Corporation.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <nbla/array.hpp>
#include <nbla/cuda/common.hpp>
#include <nbla/cuda/function/split.hpp>
#include <nbla/cuda/math.hpp>
#include <nbla/variable.hpp>

namespace nbla {

template <typename T>
__global__ void forward_split_kernel(const int num, const int num_outputs_,
                                     const int outer_size_,
                                     const int inner_size_, const int i0,
                                     const T *x, T *y) {
  NBLA_CUDA_KERNEL_LOOP(idx, num) {
    const int i1 = idx / inner_size_;
    const int i2 = idx % inner_size_;
    y[i1 * inner_size_ + i2] =
        x[i1 * (inner_size_ * num_outputs_) + i0 * inner_size_ + i2];
  }
}

template <typename T>
void SplitCuda<T>::forward_impl(const Variables &inputs,
                                const Variables &outputs) {
  cuda_set_device(std::stoi(this->ctx_.device_id));
  const Tc *x = inputs[0]->get_data_pointer<Tc>(this->ctx_);
  for (int i0 = 0; i0 < this->num_outputs_; ++i0) {
    Tc *y = outputs[i0]->cast_data_and_get_pointer<Tc>(this->ctx_, true);
    NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(
        forward_split_kernel, this->inner_size_ * this->outer_size_,
        this->num_outputs_, this->outer_size_, this->inner_size_, i0, x, y);
  }
}

template <typename T, bool accum>
__global__ void backward_split_kernel(const int num, const int num_outputs_,
                                      const int outer_size_,
                                      const int inner_size_, const int i0,
                                      T *dx, const T *dy) {
  NBLA_CUDA_KERNEL_LOOP(idx, num) {
    const int i1 = idx / inner_size_;
    const int i2 = idx % inner_size_;
    T &ref = dx[i1 * (inner_size_ * num_outputs_) + i0 * inner_size_ + i2];
    ref = (accum ? ref : (T)0) + dy[i1 * inner_size_ + i2];
  }
}

template <typename T>
void SplitCuda<T>::backward_impl(const Variables &inputs,
                                 const Variables &outputs,
                                 const vector<bool> &propagate_down,
                                 const vector<bool> &accum) {
  if (!propagate_down[0]) {
    return;
  }
  cuda_set_device(std::stoi(this->ctx_.device_id));
  Tc *dx = inputs[0]->cast_grad_and_get_pointer<Tc>(this->ctx_, !accum[0]);
  for (int i0 = 0; i0 < this->num_outputs_; ++i0) {
    const Tc *dy = outputs[i0]->get_grad_pointer<Tc>(this->ctx_);
    if (accum[0]) {
      NBLA_CUDA_LAUNCH_KERNEL_SIMPLE((backward_split_kernel<Tc, true>),
                                     this->inner_size_ * this->outer_size_,
                                     this->num_outputs_, this->outer_size_,
                                     this->inner_size_, i0, dx, dy);
    } else {
      NBLA_CUDA_LAUNCH_KERNEL_SIMPLE((backward_split_kernel<Tc, false>),
                                     this->inner_size_ * this->outer_size_,
                                     this->num_outputs_, this->outer_size_,
                                     this->inner_size_, i0, dx, dy);
    }
  }
}
}
