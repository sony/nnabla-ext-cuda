// Copyright 2021 Sony Corporation.
// Copyright 2021,2022 Sony Group Corporation.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <nbla/array.hpp>
#include <nbla/cuda/common.hpp>
#include <nbla/cuda/function/cumsum.hpp>
#include <nbla/cuda/utils/scan_ops/sum.cuh>
#include <nbla/variable.hpp>

namespace nbla {

template <typename T>
void CumSumCuda<T>::setup_impl(const Variables &inputs,
                               const Variables &outputs) {
  CumSum<T>::setup_impl(inputs, outputs);
  cuda_set_device(this->device_);

  scan_setup_forward_(inputs[0]->shape(), this->axis_, this->exclusive_,
                      this->reverse_);
  scan_setup_backward_ = scan_setup_forward_;
  scan_setup_backward_.reverse = !this->reverse_;
}

template <typename T>
void CumSumCuda<T>::forward_impl(const Variables &inputs,
                                 const Variables &outputs) {
  cuda_set_device(this->device_);
  const Tcu *x = inputs[0]->get_data_pointer<Tcu>(this->ctx_);
  Tcu *y = outputs[0]->cast_data_and_get_pointer<Tcu>(this->ctx_, true);

  device_cumsum(this->ctx_, x, y, scan_setup_forward_, false /* accum */);
}

template <typename T>
void CumSumCuda<T>::backward_impl(const Variables &inputs,
                                  const Variables &outputs,
                                  const vector<bool> &propagate_down,
                                  const vector<bool> &accum) {
  if (!(propagate_down[0])) {
    return;
  }
  cuda_set_device(this->device_);

  const Tcu *g_y = outputs[0]->get_grad_pointer<Tcu>(this->ctx_);
  Tcu *g_x = inputs[0]->cast_grad_and_get_pointer<Tcu>(this->ctx_, !accum[0]);

  // Perform reversed cumsum for `g_y`.
  // d_x = cumsum_reverse(d_y)
  device_cumsum(this->ctx_, g_y, g_x, scan_setup_backward_, accum[0]);
}
}