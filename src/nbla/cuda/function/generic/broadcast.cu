#include "hip/hip_runtime.h"
// Copyright (c) 2017 Sony Corporation. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <nbla/array.hpp>
#include <nbla/cuda/common.hpp>
#include <nbla/cuda/function/broadcast.hpp>
#include <nbla/function/sum.hpp>
#include <nbla/function/transpose.hpp>

#include <numeric>

namespace nbla {

// ----------------------------------------------------------------------------
// Forward
// ----------------------------------------------------------------------------
template <typename T>
void BroadcastCuda<T>::setup_impl(const Variables &inputs,

                                  const Variables &outputs) {
  Broadcast<T>::setup_impl(inputs, outputs);
  int ndim = outputs[0]->ndim();
  auto inshape = inputs[0]->shape();
  vector<int> broadcast_dims;
  if (inputs[0]->ndim() == 0) {
    // If input is a scalar.
    broadcast_dims.resize(this->shape_.size());
    std::iota(broadcast_dims.begin(), broadcast_dims.end(), 0);
  } else {
    for (int d = 0; d < ndim; ++d) {
      if (this->shape_[d] != inshape[d])
        broadcast_dims.push_back(d);
    }
  }
  broadcast_dims_ = broadcast_dims;
  if (broadcast_dims.size() == 0)
    return;
  f_sum_ = create_Sum(this->ctx_, /*axis*/ broadcast_dims, /*keepdims*/ true);
}

// ----------------------------------------------------------------------------
// Strided index getter
// ----------------------------------------------------------------------------
template <int ND> struct strided_index_cuda {
  static __device__ int get(int y_index, const int *stride_x,
                            const int *shape_y) {
    int stride = 1;
    int x_index = 0;
    strided_index_cuda<ND - 1>::_get(y_index, stride_x, shape_y, stride,
                                     x_index);
    return x_index;
  }
  static __device__ void _get(int y_index, const int *stride_x,
                              const int *shape_y, int &stride, int &x_index) {
    const int dim_index = int(y_index / stride) % shape_y[ND];
    stride *= shape_y[ND];
    x_index += dim_index * stride_x[ND];
    strided_index_cuda<ND - 1>::_get(y_index, stride_x, shape_y, stride,
                                     x_index);
  }
};
template <> struct strided_index_cuda<0> {
  static __device__ int get(int y_index, const int *stride_x,
                            const int *shape_y) {
    return 0;
  }
  static __device__ void _get(int y_index, const int *stride_x,
                              const int *shape_y, int &stride, int &x_index) {
    const int dim_index = int(y_index / stride) % shape_y[0];
    stride *= shape_y[0];
    x_index += dim_index * stride_x[0];
  }
};

// ----------------------------------------------------------------------------
// Broadcast kernel
// ----------------------------------------------------------------------------
template <int Ndim, typename T>
__global__ void kernel_broadcast(size_t size, const T *x, const int *stride_x,
                                 const int *shape_y, T *y) {
  NBLA_CUDA_KERNEL_LOOP(idx, size) {
    int jdx = strided_index_cuda<Ndim>::get(idx, stride_x, shape_y);
    y[idx] = x[jdx];
  }
}

// ----------------------------------------------------------------------------
// Unrolled broadcast caller for templated dimension
// ----------------------------------------------------------------------------
template <int ND, typename T> struct switch_broadcast_cuda {
  static void call(int num, size_t size, const T *x, const int *stride_x,
                   const int *shape_y, T *y) {
    if (ND == num) {
      const int blocks = NBLA_CUDA_GET_BLOCKS(size);
      const int inkernel_loop = NBLA_CEIL_INT_DIV(blocks, NBLA_CUDA_MAX_BLOCKS);
      const int total_blocks = NBLA_CEIL_INT_DIV(blocks, inkernel_loop);
      kernel_broadcast<ND, T><<<total_blocks, NBLA_CUDA_NUM_THREADS>>>(
          size, x, stride_x, shape_y, y);
      NBLA_CUDA_KERNEL_CHECK();
      return;
    }
    switch_broadcast_cuda<ND - 1, T>::call(num, size, x, stride_x, shape_y, y);
  }
};

template <typename T> struct switch_broadcast_cuda<-1, T> {
  static void call(int num, size_t size, const T *x, const int *stride_x,
                   const int *shape_y, T *y) {
    NBLA_ERROR(error_code::not_implemented,
               "Broadcast is not implemented for %d dimensional array.", num);
  }
};

// ----------------------------------------------------------------------------
// Forward
// ----------------------------------------------------------------------------
template <typename T>
void BroadcastCuda<T>::forward_impl(const Variables &inputs,
                                    const Variables &outputs) {
  const Tc *x = inputs[0]->get_data_pointer<Tc>(this->ctx_);
  Tc *y = outputs[0]->cast_data_and_get_pointer<Tc>(this->ctx_, true);
  auto _iarr = [this](Variable &v) {
    return v.get_data_pointer<int>(this->ctx_);
  };
  const int *stride_x = _iarr(this->stride_x_);
  const int *shape_y = _iarr(this->shape_y_);
  int ndim = inputs[0]->ndim();
  int size = outputs[0]->size();
  cuda_set_device(device_);
  switch_broadcast_cuda<NBLA_BROADCAST_MAX_DIM, Tc>::call(ndim, size, x,
                                                          stride_x, shape_y, y);
}

template <typename T>
__global__ void kernel_add_grad(int size, const T *g, T *dx) {
  NBLA_CUDA_KERNEL_LOOP(idx, size) { dx[idx] += g[idx]; }
}

template <typename T>
void BroadcastCuda<T>::backward_impl(const Variables &inputs,
                                     const Variables &outputs,
                                     const vector<bool> &propagate_down,
                                     const vector<bool> &accum) {
  if (!propagate_down[0])
    return;
  shared_ptr<Variable> sum_input = make_shared<Variable>(outputs[0]->grad());
  shared_ptr<Variable> sum_output;
  if (f_sum_) {
    if (!accum[0]) {
      sum_output = make_shared<Variable>(inputs[0]->grad());
      f_sum_->setup(Variables{sum_input.get()}, Variables{sum_output.get()});
      f_sum_->forward(Variables{sum_input.get()}, Variables{sum_output.get()});
      return;
    }
    sum_output = make_shared<Variable>(inputs[0]->shape());
    f_sum_->setup(Variables{sum_input.get()}, Variables{sum_output.get()});
    f_sum_->forward(Variables{sum_input.get()}, Variables{sum_output.get()});
  } else {
    if (!accum[0])
      inputs[0]->grad()->zero();
  }
  auto _get = [this](Variable *v) {
    return v->get_data_pointer<Tc>(this->ctx_);
  };
  auto _gget = [this](Variable *v) {
    return v->get_grad_pointer<Tc>(this->ctx_);
  };
  cuda_set_device(device_);
  const Tc *g = f_sum_ ? _get(sum_output.get()) : _gget(outputs[0]);
  Tc *dx = inputs[0]->cast_grad_and_get_pointer<Tc>(this->ctx_, false);
  NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_add_grad, inputs[0]->size(), g, dx);
}
}
