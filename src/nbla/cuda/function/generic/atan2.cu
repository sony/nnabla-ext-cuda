#include "hip/hip_runtime.h"
// Copyright (c) 2017 Sony Corporation. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

// atan2.cpp

#include <nbla/cuda/function/atan2.hpp>
#include <nbla/cuda/function/utils/base_transform_binary.cuh>

#include <cmath>

namespace nbla {

NBLA_DEFINE_TRANSFORM_BINARY_CUDA(ATan2, std::atan2(x0, x1),
                                  dy *x1 / (x0 * x0 + x1 * x1),
                                  -dy *x0 / (x0 * x0 + x1 * x1));
}
