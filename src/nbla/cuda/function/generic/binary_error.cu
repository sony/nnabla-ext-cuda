#include "hip/hip_runtime.h"
// Copyright 2018,2019,2020,2021 Sony Corporation.
// Copyright 2022 Sony Group Corporation.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <nbla/array.hpp>
#include <nbla/cuda/common.hpp>
#include <nbla/cuda/function/binary_error.hpp>
#include <nbla/cuda/math.hpp>
#include <nbla/variable.hpp>

namespace nbla {

template <typename T>
__global__ void kernel_binary_error_forward(const int size, const T *x0,
                                            const T *x1, T *y) {
  NBLA_CUDA_KERNEL_LOOP(s, size) { y[s] = (x0[s] >= 0.5) != (x1[s] >= 0.5); }
}

template <typename T>
void BinaryErrorCuda<T>::forward_impl(const Variables &inputs,
                                      const Variables &outputs) {
  cuda_set_device(std::stoi(this->ctx_.device_id));
  const Tc *x0 = inputs[0]->get_data_pointer<Tc>(this->ctx_);
  const Tc *x1 = inputs[1]->get_data_pointer<Tc>(this->ctx_);
  Tc *y = outputs[0]->cast_data_and_get_pointer<Tc>(this->ctx_, true);
  const Size_t size = inputs[0]->size();
  NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_binary_error_forward, size, x0, x1, y);
}

template <typename T>
void BinaryErrorCuda<T>::backward_impl(const Variables &inputs,
                                       const Variables &outputs,
                                       const vector<bool> &propagate_down,
                                       const vector<bool> &accum) {
  // not supported
}
}
