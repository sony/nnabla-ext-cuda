#include "hip/hip_runtime.h"
// Copyright 2021,2022 Sony Group Corporation.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <nbla/array.hpp>
#include <nbla/cuda/common.hpp>
#include <nbla/cuda/function/roi_align.hpp>
#include <nbla/cuda/utils/atomic_add.cuh>
#include <nbla/variable.hpp>

namespace nbla {

namespace {
template <typename T> struct Box { T batch_index, x1, y1, x2, y2; };

template <typename T>
__forceinline__ __device__ T clamp(const T x, const T low, const T high) {
  return max(low, min(high, x));
}

template <typename T>
__forceinline__ __device__ int sampling_grid(const int sampling_ratio,
                                             const T step_size) {
  return sampling_ratio > 0 ? sampling_ratio
                            : static_cast<int>(ceil(step_size));
}
}

template <typename T, typename SIZE_T>
__global__ void roi_align_forward_kernel_nchw(
    const SIZE_T size, const T *input_data, const T *boxes_data, T *output_data,
    const SIZE_T samples, const SIZE_T channels, const SIZE_T input_rows,
    const SIZE_T input_cols, const SIZE_T input_stride_c,
    const SIZE_T input_stride_n, const SIZE_T output_rows,
    const SIZE_T output_cols, const SIZE_T output_stride_c,
    const SIZE_T output_stride_n, const int sampling_ratio,
    const float spatial_scale_y, const float spatial_scale_x) {
  NBLA_CUDA_KERNEL_LOOP(output_index, size) {
    SIZE_T n = output_index / output_stride_n;
    SIZE_T index = output_index - n * output_stride_n;
    SIZE_T c = index / output_stride_c;
    index -= c * output_stride_c;
    SIZE_T y = index / output_cols;
    SIZE_T x = index - y * output_cols;

    auto roi = *reinterpret_cast<Box<T> const *>(boxes_data + n * 5);
    auto const roi_x1 = roi.x1 * spatial_scale_x - 0.5f;
    auto const roi_y1 = roi.y1 * spatial_scale_y - 0.5f;
    auto const roi_x2 = roi.x2 * spatial_scale_x - 0.5f;
    auto const roi_y2 = roi.y2 * spatial_scale_y - 0.5f;
    auto const roi_index = clamp<SIZE_T>(roi.batch_index, 0, samples - 1);

    auto const step_size_x = (roi_x2 - roi_x1) / output_cols;
    auto const step_size_y = (roi_y2 - roi_y1) / output_rows;

    auto const grid_size_x = sampling_grid(sampling_ratio, step_size_x);
    auto const grid_size_y = sampling_grid(sampling_ratio, step_size_y);

    auto const step_size_xx = step_size_x / grid_size_x;
    auto const step_size_yy = step_size_y / grid_size_y;
    auto const grid_size_xy = grid_size_x * grid_size_y;

    auto const half_step_xx = 0.5f * step_size_xx;
    auto const half_step_yy = 0.5f * step_size_yy;

    auto const xf = roi_x1 + x * step_size_x + half_step_xx;
    auto const yf = roi_y1 + y * step_size_y + half_step_yy;

    auto input_sample_data = input_data + roi_index * input_stride_n;
    auto input_channel_data = input_sample_data + c * input_stride_c;
    auto output_value = 0.0f;

    for (auto yy = 0; yy < grid_size_y; yy++) {
      auto yyf = yf + yy * step_size_yy;

      if (yyf < -1.0f || yyf > input_rows)
        continue;

      yyf = clamp<T>(yyf, 0, input_rows - 1);
      auto const y_lo = static_cast<SIZE_T>(yyf);
      auto const y_hi = min(y_lo + 1, input_rows - 1);
      auto const ly = yyf - floor(yyf);
      auto const hy = 1.0f - ly;

      for (auto xx = 0; xx < grid_size_x; xx++) {
        auto xxf = xf + xx * step_size_xx;

        if (xxf < -1.0f || xxf > input_cols)
          continue;

        xxf = clamp<T>(xxf, 0, input_cols - 1);
        auto const x_lo = static_cast<SIZE_T>(xxf);
        auto const x_hi = min(x_lo + 1, input_cols - 1);
        auto const lx = xxf - floor(xxf);
        auto const hx = 1.0f - lx;

        auto const p1 = y_lo * input_cols + x_lo;
        auto const p2 = y_lo * input_cols + x_hi;
        auto const p3 = y_hi * input_cols + x_lo;
        auto const p4 = y_hi * input_cols + x_hi;
        output_value += hy * hx * input_channel_data[p1];
        output_value += hy * lx * input_channel_data[p2];
        output_value += ly * hx * input_channel_data[p3];
        output_value += ly * lx * input_channel_data[p4];
      }
    }
    output_data[output_index] = static_cast<T>(output_value / grid_size_xy);
  }
}

template <typename T, typename SIZE_T>
__global__ void roi_align_forward_kernel_nhwc(
    const SIZE_T size, const T *input_data, const T *boxes_data, T *output_data,
    const SIZE_T samples, const SIZE_T channels, const SIZE_T input_rows,
    const SIZE_T input_cols, const SIZE_T input_stride_n,
    const SIZE_T output_rows, const SIZE_T output_cols,
    const SIZE_T output_size_xy, const int sampling_ratio,
    const float spatial_scale_y, const float spatial_scale_x) {
  NBLA_CUDA_KERNEL_LOOP(thread_index, size) {
    SIZE_T n = thread_index / output_size_xy;
    SIZE_T i = thread_index - n * output_size_xy;
    SIZE_T y = i / output_cols;
    SIZE_T x = i - y * output_cols;

    auto roi = *reinterpret_cast<Box<T> const *>(boxes_data + n * 5);
    auto const roi_x1 = roi.x1 * spatial_scale_x - 0.5f;
    auto const roi_y1 = roi.y1 * spatial_scale_y - 0.5f;
    auto const roi_x2 = roi.x2 * spatial_scale_x - 0.5f;
    auto const roi_y2 = roi.y2 * spatial_scale_y - 0.5f;
    auto const roi_index = clamp<SIZE_T>(roi.batch_index, 0, samples - 1);

    auto const step_size_x = (roi_x2 - roi_x1) / output_cols;
    auto const step_size_y = (roi_y2 - roi_y1) / output_rows;

    auto const grid_size_x = sampling_grid(sampling_ratio, step_size_x);
    auto const grid_size_y = sampling_grid(sampling_ratio, step_size_y);

    auto const step_size_xx = step_size_x / grid_size_x;
    auto const step_size_yy = step_size_y / grid_size_y;
    auto const grid_size_xy = grid_size_x * grid_size_y;

    auto const half_step_xx = 0.5f * step_size_xx;
    auto const half_step_yy = 0.5f * step_size_yy;

    auto const xf = roi_x1 + x * step_size_x + half_step_xx;
    auto const yf = roi_y1 + y * step_size_y + half_step_yy;

    auto input_sample_data = input_data + roi_index * input_stride_n;
    auto output_channel_data = output_data + thread_index * channels;

    for (auto c = 0; c < channels; c++) {
      output_channel_data[c] = T(0);
    }

    for (auto yy = 0; yy < grid_size_y; yy++) {
      auto yyf = yf + yy * step_size_yy;

      if (yyf < -1.0f || yyf > input_rows)
        continue;

      yyf = clamp<T>(yyf, 0, input_rows - 1);
      auto const y_lo = static_cast<SIZE_T>(yyf);
      auto const y_hi = min(y_lo + 1, input_rows - 1);
      auto const ly = yyf - floor(yyf);
      auto const hy = 1.0f - ly;

      for (auto xx = 0; xx < grid_size_x; xx++) {
        auto xxf = xf + xx * step_size_xx;

        if (xxf < -1.0f || xxf > input_cols)
          continue;

        xxf = clamp<T>(xxf, 0, input_cols - 1);
        auto const x_lo = static_cast<SIZE_T>(xxf);
        auto const x_hi = min(x_lo + 1, input_cols - 1);
        auto const lx = xxf - floor(xxf);
        auto const hx = 1.0f - lx;

        auto const p1 = (y_lo * input_cols + x_lo) * channels;
        auto const p2 = (y_lo * input_cols + x_hi) * channels;
        auto const p3 = (y_hi * input_cols + x_lo) * channels;
        auto const p4 = (y_hi * input_cols + x_hi) * channels;
        for (auto c = 0; c < channels; c++) {
          auto output_data_value = 0.0f;
          output_data_value += hy * hx * input_sample_data[p1 + c];
          output_data_value += hy * lx * input_sample_data[p2 + c];
          output_data_value += ly * hx * input_sample_data[p3 + c];
          output_data_value += ly * lx * input_sample_data[p4 + c];
          output_channel_data[c] += static_cast<T>(output_data_value);
        }
      }
    }
    for (auto c = 0; c < channels; c++) {
      output_channel_data[c] /= grid_size_xy;
    }
  }
}

template <typename T, typename SIZE_T>
__global__ void roi_align_backward_kernel_nchw(
    const SIZE_T size, T *input_grad, const T *boxes_data, const T *output_grad,
    const SIZE_T samples, const SIZE_T channels, const SIZE_T input_rows,
    const SIZE_T input_cols, const SIZE_T input_channel_size,
    const SIZE_T input_sample_size, const SIZE_T output_rows,
    const SIZE_T output_cols, const SIZE_T output_stride_c,
    const SIZE_T output_stride_n, const int sampling_ratio,
    const float spatial_scale_y, const float spatial_scale_x) {
  NBLA_CUDA_KERNEL_LOOP(output_index, size) {
    SIZE_T n = output_index / output_stride_n;
    SIZE_T index = output_index - n * output_stride_n;
    SIZE_T c = index / output_stride_c;
    index -= c * output_stride_c;
    SIZE_T y = index / output_cols;
    SIZE_T x = index - y * output_cols;

    auto roi = *reinterpret_cast<Box<T> const *>(boxes_data + n * 5);
    auto const roi_x1 = roi.x1 * spatial_scale_x - 0.5f;
    auto const roi_y1 = roi.y1 * spatial_scale_y - 0.5f;
    auto const roi_x2 = roi.x2 * spatial_scale_x - 0.5f;
    auto const roi_y2 = roi.y2 * spatial_scale_y - 0.5f;
    auto const roi_index = clamp<SIZE_T>(roi.batch_index, 0, samples - 1);

    auto const step_size_x = (roi_x2 - roi_x1) / output_cols;
    auto const step_size_y = (roi_y2 - roi_y1) / output_rows;

    auto const grid_size_x = sampling_grid(sampling_ratio, step_size_x);
    auto const grid_size_y = sampling_grid(sampling_ratio, step_size_y);

    auto const step_size_xx = step_size_x / grid_size_x;
    auto const step_size_yy = step_size_y / grid_size_y;
    auto const grid_size_xy = grid_size_x * grid_size_y;

    auto const half_step_xx = 0.5f * step_size_xx;
    auto const half_step_yy = 0.5f * step_size_yy;

    auto const xf = roi_x1 + x * step_size_x + half_step_xx;
    auto const yf = roi_y1 + y * step_size_y + half_step_yy;

    auto input_sample_grad = input_grad + roi_index * input_sample_size;
    auto input_channel_grad = input_sample_grad + c * input_channel_size;
    auto output_grad_value = output_grad[output_index] / grid_size_xy;

    for (auto yy = 0; yy < grid_size_y; yy++) {
      auto yyf = yf + yy * step_size_yy;

      if (yyf < -1.0f || yyf > input_rows)
        continue;

      yyf = clamp<T>(yyf, 0, input_rows - 1);
      auto const y_lo = static_cast<SIZE_T>(yyf);
      auto const y_hi = min(y_lo + 1, input_rows - 1);
      auto const ly = yyf - floor(yyf);
      auto const hy = 1.0f - ly;

      for (auto xx = 0; xx < grid_size_x; xx++) {
        auto xxf = xf + xx * step_size_xx;

        if (xxf < -1.0f || xxf > input_cols)
          continue;

        xxf = clamp<T>(xxf, 0, input_cols - 1);
        auto const x_lo = static_cast<SIZE_T>(xxf);
        auto const x_hi = min(x_lo + 1, input_cols - 1);
        auto const lx = xxf - floor(xxf);
        auto const hx = 1.0f - lx;

        auto const p1 = y_lo * input_cols + x_lo;
        auto const p2 = y_lo * input_cols + x_hi;
        auto const p3 = y_hi * input_cols + x_lo;
        auto const p4 = y_hi * input_cols + x_hi;
        const T v1 = static_cast<T>(hy * hx * output_grad_value);
        const T v2 = static_cast<T>(hy * lx * output_grad_value);
        const T v3 = static_cast<T>(ly * hx * output_grad_value);
        const T v4 = static_cast<T>(ly * lx * output_grad_value);
        atomic_add(&input_channel_grad[p1], v1);
        atomic_add(&input_channel_grad[p2], v2);
        atomic_add(&input_channel_grad[p3], v3);
        atomic_add(&input_channel_grad[p4], v4);
      }
    }
  }
}

template <typename T, typename SIZE_T>
__global__ void roi_align_backward_kernel_nhwc(
    const SIZE_T size, T *input_grad, const T *boxes_data, const T *output_grad,
    const SIZE_T samples, const SIZE_T channels, const SIZE_T input_rows,
    const SIZE_T input_cols, const SIZE_T input_stride_n,
    const SIZE_T output_rows, const SIZE_T output_cols,
    const SIZE_T output_size_xy, const int sampling_ratio,
    const float spatial_scale_y, const float spatial_scale_x) {
  NBLA_CUDA_KERNEL_LOOP(thread_index, size) {
    SIZE_T n = thread_index / output_size_xy;
    SIZE_T i = thread_index - n * output_size_xy;
    SIZE_T y = i / output_cols;
    SIZE_T x = i - y * output_cols;

    auto roi = *reinterpret_cast<Box<T> const *>(boxes_data + n * 5);
    auto const roi_x1 = roi.x1 * spatial_scale_x - 0.5f;
    auto const roi_y1 = roi.y1 * spatial_scale_y - 0.5f;
    auto const roi_x2 = roi.x2 * spatial_scale_x - 0.5f;
    auto const roi_y2 = roi.y2 * spatial_scale_y - 0.5f;
    auto const roi_index = clamp<SIZE_T>(roi.batch_index, 0, samples - 1);

    auto const step_size_x = (roi_x2 - roi_x1) / output_cols;
    auto const step_size_y = (roi_y2 - roi_y1) / output_rows;

    auto const grid_size_x = sampling_grid(sampling_ratio, step_size_x);
    auto const grid_size_y = sampling_grid(sampling_ratio, step_size_y);

    auto const step_size_xx = step_size_x / grid_size_x;
    auto const step_size_yy = step_size_y / grid_size_y;
    auto const grid_size_xy = grid_size_x * grid_size_y;

    auto const half_step_xx = 0.5f * step_size_xx;
    auto const half_step_yy = 0.5f * step_size_yy;

    auto const xf = roi_x1 + x * step_size_x + half_step_xx;
    auto const yf = roi_y1 + y * step_size_y + half_step_yy;

    auto input_sample_grad = input_grad + roi_index * input_stride_n;
    auto output_channel_grad = output_grad + thread_index * channels;

    for (auto yy = 0; yy < grid_size_y; yy++) {
      auto yyf = yf + static_cast<T>(yy) * step_size_yy;

      if (yyf < -1.0f || yyf > static_cast<T>(input_rows))
        continue;

      yyf = clamp<T>(yyf, 0, input_rows - 1);
      auto const y_lo = static_cast<SIZE_T>(yyf);
      auto const y_hi = min(y_lo + 1, input_rows - 1);
      auto const ly = yyf - floor(yyf);
      auto const hy = 1.0f - ly;

      for (auto xx = 0; xx < grid_size_x; xx++) {
        auto xxf = xf + static_cast<T>(xx) * step_size_xx;

        if (xxf < -1.0f || xxf > static_cast<T>(input_cols))
          continue;

        xxf = clamp<T>(xxf, 0, input_cols - 1);
        auto const x_lo = static_cast<SIZE_T>(xxf);
        auto const x_hi = min(x_lo + 1, input_cols - 1);
        auto const lx = xxf - floor(xxf);
        auto const hx = 1.0f - lx;

        auto const p1 = (y_lo * input_cols + x_lo) * channels;
        auto const p2 = (y_lo * input_cols + x_hi) * channels;
        auto const p3 = (y_hi * input_cols + x_lo) * channels;
        auto const p4 = (y_hi * input_cols + x_hi) * channels;
        for (auto c = 0; c < channels; c++) {
          auto const grad_value = static_cast<float>(output_channel_grad[c]);
          const T v1 = static_cast<T>(hy * hx * grad_value / grid_size_xy);
          const T v2 = static_cast<T>(hy * lx * grad_value / grid_size_xy);
          const T v3 = static_cast<T>(ly * hx * grad_value / grid_size_xy);
          const T v4 = static_cast<T>(ly * lx * grad_value / grid_size_xy);
          atomic_add(&input_sample_grad[p1 + c], v1);
          atomic_add(&input_sample_grad[p2 + c], v2);
          atomic_add(&input_sample_grad[p3 + c], v3);
          atomic_add(&input_sample_grad[p4 + c], v4);
        }
      }
    }
  }
}

template <typename T>
void RoiAlignCuda<T>::setup_impl(const Variables &inputs,
                                 const Variables &outputs) {
  RoiAlign<T>::setup_impl(inputs, outputs);
  cuda_set_device(this->device_);
}

template <typename T>
void RoiAlignCuda<T>::forward_impl(const Variables &inputs,
                                   const Variables &outputs) {
  cuda_set_device(this->device_);

  auto input = inputs.at(0);
  auto boxes = inputs.at(1);
  auto output = outputs.at(0);

  auto input_data = input->get_data_pointer<Tcu>(this->ctx_);
  auto boxes_data = boxes->get_data_pointer<Tcu>(this->ctx_);
  auto output_data = output->cast_data_and_get_pointer<Tcu>(this->ctx_, true);

  auto const samples = input->shape().at(0);
  auto const spatial_scale_y = this->spatial_scale_.at(0);
  auto const spatial_scale_x = this->spatial_scale_.at(1);

  if (!this->channel_last_) {
    auto const channels = input->shape().at(1);
    auto const input_rows = input->shape().at(2);
    auto const input_cols = input->shape().at(3);
    auto const output_rows = output->shape().at(2);
    auto const output_cols = output->shape().at(3);
    auto const input_stride_n = input->strides().at(0);
    auto const input_stride_c = input->strides().at(1);
    auto const output_stride_n = output->strides().at(0);
    auto const output_stride_c = output->strides().at(1);
    auto const nthreads = output->size();

    if (output->size() <= INT32_MAX) {
      auto kernel = roi_align_forward_kernel_nchw<Tcu, int32_t>;

      NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(
          kernel, nthreads, input_data, boxes_data, output_data, samples,
          channels, input_rows, input_cols, input_stride_c, input_stride_n,
          output_rows, output_cols, output_stride_c, output_stride_n,
          this->sampling_ratio_, spatial_scale_y, spatial_scale_x);
    } else {
      auto kernel = roi_align_forward_kernel_nchw<Tcu, int64_t>;

      NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(
          kernel, nthreads, input_data, boxes_data, output_data, samples,
          channels, input_rows, input_cols, input_stride_c, input_stride_n,
          output_rows, output_cols, output_stride_c, output_stride_n,
          this->sampling_ratio_, spatial_scale_y, spatial_scale_x);
    }
  } else {
    auto const channels = input->shape().at(3);
    auto const input_rows = input->shape().at(1);
    auto const input_cols = input->shape().at(2);
    auto const output_rows = output->shape().at(1);
    auto const output_cols = output->shape().at(2);
    auto const input_stride_n = input->strides().at(0);
    auto const output_size_xy = output_rows * output_cols;
    auto const nthreads = output->size() / channels;

    if (output->size() <= INT32_MAX) {
      auto kernel = roi_align_forward_kernel_nhwc<Tcu, int32_t>;

      NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(
          kernel, nthreads, input_data, boxes_data, output_data, samples,
          channels, input_rows, input_cols, input_stride_n, output_rows,
          output_cols, output_size_xy, this->sampling_ratio_, spatial_scale_y,
          spatial_scale_x);
    } else {
      auto kernel = roi_align_forward_kernel_nhwc<Tcu, int64_t>;

      NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(
          kernel, nthreads, input_data, boxes_data, output_data, samples,
          channels, input_rows, input_cols, input_stride_n, output_rows,
          output_cols, output_size_xy, this->sampling_ratio_, spatial_scale_y,
          spatial_scale_x);
    }
  }
}

template <typename T>
void RoiAlignCuda<T>::backward_impl(const Variables &inputs,
                                    const Variables &outputs,
                                    const vector<bool> &propagate_down,
                                    const vector<bool> &accum) {

  if (!propagate_down[0]) {
    return;
  }
  cuda_set_device(this->device_);

  auto input = inputs.at(0);
  auto boxes = inputs.at(1);
  auto output = outputs.at(0);

  auto input_grad = input->cast_grad_and_get_pointer<Tcu>(this->ctx_, false);
  auto boxes_data = boxes->get_data_pointer<Tcu>(this->ctx_);
  auto output_grad = output->get_grad_pointer<Tcu>(this->ctx_);

  auto const samples = input->shape().at(0);
  auto const spatial_scale_y = this->spatial_scale_.at(0);
  auto const spatial_scale_x = this->spatial_scale_.at(1);

  if (!this->channel_last_) {
    auto const channels = input->shape().at(1);
    auto const input_rows = input->shape().at(2);
    auto const input_cols = input->shape().at(3);
    auto const output_rows = output->shape().at(2);
    auto const output_cols = output->shape().at(3);
    auto const input_stride_n = input->strides().at(0);
    auto const input_stride_c = input->strides().at(1);
    auto const output_stride_n = output->strides().at(0);
    auto const output_stride_c = output->strides().at(1);
    auto const nthreads = output->size();

    if (output->size() <= INT32_MAX) {
      auto kernel = roi_align_backward_kernel_nchw<Tcu, int32_t>;

      NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(
          kernel, nthreads, input_grad, boxes_data, output_grad, samples,
          channels, input_rows, input_cols, input_stride_c, input_stride_n,
          output_rows, output_cols, output_stride_c, output_stride_n,
          this->sampling_ratio_, spatial_scale_y, spatial_scale_x);
    } else {
      auto kernel = roi_align_backward_kernel_nchw<Tcu, int64_t>;

      NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(
          kernel, nthreads, input_grad, boxes_data, output_grad, samples,
          channels, input_rows, input_cols, input_stride_c, input_stride_n,
          output_rows, output_cols, output_stride_c, output_stride_n,
          this->sampling_ratio_, spatial_scale_y, spatial_scale_x);
    }
  } else {
    auto const channels = input->shape().at(3);
    auto const input_rows = input->shape().at(1);
    auto const input_cols = input->shape().at(2);
    auto const output_rows = output->shape().at(1);
    auto const output_cols = output->shape().at(2);
    auto const input_stride_n = input->strides().at(0);
    auto const output_size_xy = output_rows * output_cols;
    auto const nthreads = output->size() / channels;

    if (output->size() <= INT32_MAX) {
      auto kernel = roi_align_backward_kernel_nhwc<Tcu, int32_t>;

      NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(
          kernel, nthreads, input_grad, boxes_data, output_grad, samples,
          channels, input_rows, input_cols, input_stride_n, output_rows,
          output_cols, output_size_xy, this->sampling_ratio_, spatial_scale_y,
          spatial_scale_x);
    } else {
      auto kernel = roi_align_backward_kernel_nhwc<Tcu, int64_t>;

      NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(
          kernel, nthreads, input_grad, boxes_data, output_grad, samples,
          channels, input_rows, input_cols, input_stride_n, output_rows,
          output_cols, output_size_xy, this->sampling_ratio_, spatial_scale_y,
          spatial_scale_x);
    }
  }
}
}
