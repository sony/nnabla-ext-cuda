// Copyright (c) 2017 Sony Corporation. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

// convolution.cu

#include <nbla/array.hpp>
#include <nbla/cuda/common.hpp>
#include <nbla/cuda/function/convolution.hpp>
#include <nbla/cuda/math.hpp>
#include <nbla/singleton_manager.hpp>
#include <nbla/variable.hpp>

#include <nbla/cuda/utils/col2im.hpp>
#include <nbla/cuda/utils/im2col.hpp>

#include <algorithm>

namespace nbla {

template <typename T>
void ConvolutionCuda<T>::setup_impl(const Variables &inputs,
                                    const Variables &outputs) {
  Convolution<T>::setup_impl(inputs, outputs);
}

template <class T>
void ConvolutionCuda<T>::forward_impl(const Variables &inputs,
                                      const Variables &outputs) {
  NBLA_CHECK(!this->channel_last_, error_code::value,
             "The passed argument channel_last_=true is not supported in CUDA "
             "Convolution.");

  cuda_set_device(std::stoi(this->ctx_.device_id));
  // Getting variable pointers
  const Tc *x = inputs[0]->get_data_pointer<Tc>(this->ctx_);
  const Tc *w = inputs[1]->get_data_pointer<Tc>(this->ctx_);
  Variable *vcol = &this->col_;
  Tc *col = vcol->cast_data_and_get_pointer<Tc>(this->ctx_, true);
  Tc *y = outputs[0]->cast_data_and_get_pointer<Tc>(this->ctx_, true);
  const Tc *b;
  if (inputs.size() == 3) {
    b = inputs[2]->get_data_pointer<Tc>(this->ctx_);
  }
  // Sample loop
  for (int n = 0; n < this->outer_size_; ++n) {
    // Im2col
    if (this->spatial_dims_ == 2) {
      im2col_cuda<Tc>(x + n * this->inner_size_i_, this->channels_i_,
                      this->spatial_shape_i_.data(), this->kernel_.data(),
                      this->pad_.data(), this->stride_.data(),
                      this->dilation_.data(), col);
    } else {
      im2col_nd_cuda<Tc>(x + n * this->inner_size_i_, this->channels_i_,
                         this->spatial_dims_, this->spatial_shape_i_.data(),
                         this->kernel_.data(), this->pad_.data(),
                         this->stride_.data(), this->dilation_.data(), col);
    }
    // Convolution by matrix multiplication
    Tc *y_n = y + n * this->inner_size_o_;
    for (int g = 0; g < this->group_; ++g) {
      // y = x * w
      cuda_gemm<Tc>(device_, y_n + g * this->row_y_ * this->col_y_, false,
                    col + g * this->row_col_ * this->col_col_, this->col_col_,
                    this->row_col_, false, w + g * this->row_w_ * this->col_w_,
                    this->col_w_, this->row_w_, false, 1, 0);
    }
    // Adding bias
    if (inputs.size() == 3) {
      const Tc *ones =
          static_cast<const Tc *>(SingletonManager::get<NNabla>()->ones(
              this->col_y_, get_dtype<Tc>(), this->ctx_));
      // y = 1s * b^T + y
      cuda_gemm<Tc>(device_, y_n, false, ones, 1, this->col_y_, true, b,
                    this->channels_o_, 1, true, 1, 1);
    }
  }
}

template <class T>
void ConvolutionCuda<T>::backward_impl(const Variables &inputs,
                                       const Variables &outputs,
                                       const vector<bool> &propagate_down,
                                       const vector<bool> &accum) {
  if (!(propagate_down[0] || propagate_down[1] ||
        (inputs.size() == 3 && propagate_down[2]))) {
    return;
  }

  NBLA_CHECK(!this->channel_last_, error_code::value,
             "The passed argument channel_last_=true is not supported in CUDA "
             "Convolution.");

  cuda_set_device(std::stoi(this->ctx_.device_id));
  const Tc *dy = outputs[0]->get_grad_pointer<Tc>(this->ctx_);
  const Tc *x;
  const Tc *w;
  Tc *dx, *dw, *db, *col;
  Variable *temp_col = &this->col_;
  if (propagate_down[0] || propagate_down[1]) {
    col = temp_col->cast_data_and_get_pointer<Tc>(this->ctx_, true);
  }
  if (propagate_down[0]) {
    if (!accum[0])
      inputs[0]->grad()->zero();
    w = inputs[1]->get_data_pointer<Tc>(this->ctx_);
    dx = inputs[0]->cast_grad_and_get_pointer<Tc>(this->ctx_, false);
  }
  if (propagate_down[1]) {
    if (!accum[1])
      inputs[1]->grad()->zero();
    x = inputs[0]->get_data_pointer<Tc>(this->ctx_);
    dw = inputs[1]->cast_grad_and_get_pointer<Tc>(this->ctx_, false);
  }
  if (inputs.size() == 3 && propagate_down[2]) {
    if (!accum[2])
      inputs[2]->grad()->zero();
    db = inputs[2]->cast_grad_and_get_pointer<Tc>(this->ctx_, false);
  }
  // Sample loop
  for (int n = 0; n < this->outer_size_; ++n) {
    const Tc *dy_n = dy + n * this->inner_size_o_;
    if (propagate_down[0]) {
      // Backprop to image
      Tc *dx_n = dx + n * this->inner_size_i_;
      for (int g = 0; g < this->group_; ++g) {
        // dx = w^T * dy
        cuda_gemm<Tc>(device_, col + this->row_col_ * this->col_col_ * g, true,
                      w + this->row_w_ * this->col_w_ * g, this->col_w_,
                      this->row_w_, false,
                      dy_n + this->row_y_ * this->col_y_ * g, this->col_y_,
                      this->row_y_, true, 1, 0);
      }
      // col2im
      if (this->spatial_dims_ == 2) {
        col2im_cuda<Tc>(col, this->channels_i_, this->spatial_shape_i_.data(),
                        this->kernel_.data(), this->pad_.data(),
                        this->stride_.data(), this->dilation_.data(), dx_n);
      } else {
        col2im_nd_cuda<Tc>(col, this->channels_i_, this->spatial_dims_,
                           this->spatial_shape_i_.data(), this->kernel_.data(),
                           this->pad_.data(), this->stride_.data(),
                           this->dilation_.data(), dx_n);
      }
    }
    if (propagate_down[1]) {
      // Backprop to weights
      // im2col
      if (this->spatial_dims_ == 2) {
        im2col_cuda<Tc>(x + n * this->inner_size_i_, this->channels_i_,
                        this->spatial_shape_i_.data(), this->kernel_.data(),
                        this->pad_.data(), this->stride_.data(),
                        this->dilation_.data(), col);
      } else {
        im2col_nd_cuda<Tc>(x + n * this->inner_size_i_, this->channels_i_,
                           this->spatial_dims_, this->spatial_shape_i_.data(),
                           this->kernel_.data(), this->pad_.data(),
                           this->stride_.data(), this->dilation_.data(), col);
      }
      // Weight convolution by matrix multiplication
      for (int g = 0; g < this->group_; ++g) {
        // dw += dy * col^T
        cuda_gemm<Tc>(device_, dw + g * this->row_w_ * this->col_w_, true,
                      dy_n + g * this->row_y_ * this->col_y_, this->col_y_,
                      this->row_y_, true,
                      col + g * this->row_col_ * this->col_col_, this->col_col_,
                      this->row_col_, false, 1, 1);
      }
    }
    if (inputs.size() == 3 && propagate_down[2]) {
      // Backprop to bias
      const Tc *ones =
          static_cast<const Tc *>(SingletonManager::get<NNabla>()->ones(
              this->col_y_, get_dtype<Tc>(), this->ctx_));
      cuda_gemv<Tc>(device_, db, dy_n, this->col_y_, this->channels_o_, true,
                    ones, this->col_y_, 1, 1);
    }
  }
}
}
