#include "hip/hip_runtime.h"
// Copyright 2018,2019,2020,2021 Sony Corporation.
// Copyright 2022 Sony Group Corporation.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

// reduce_sum.cu

#include <algorithm>
#include <nbla/array.hpp>
#include <nbla/cuda/common.hpp>
#include <nbla/cuda/function/reduce_sum.hpp>
#include <nbla/variable.hpp>

#include <thrust/copy.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>

namespace nbla {

template <typename T>
__global__ void kernel_reduce_sum_backward(const int num, T *dx, const T *dy) {
  NBLA_CUDA_KERNEL_LOOP(idx, num) { dx[idx] += *dy; }
}

template <class T>
void ReduceSumCuda<T>::forward_impl(const Variables &inputs,
                                    const Variables &outputs) {
  typedef typename CudaTypeForceFloat<T>::type Tc;
  cuda_set_device(std::stoi(this->ctx_.device_id));
  Tc *y = outputs[0]->cast_data_and_get_pointer<Tc>(this->ctx_, true);
  const Size_t size = inputs[0]->size();
  thrust::device_ptr<const Tc> x(inputs[0]->get_data_pointer<Tc>(this->ctx_));
  Tc sum = thrust::reduce(x, x + size, (Tc)0, thrust::plus<Tc>());
  hipMemcpy(y, &sum, sizeof(Tc), hipMemcpyHostToDevice);
}

template <class T>
void ReduceSumCuda<T>::backward_impl(const Variables &inputs,
                                     const Variables &outputs,
                                     const vector<bool> &propagate_down,
                                     const vector<bool> &accum) {
  if (!propagate_down[0]) {
    return;
  }
  cuda_set_device(std::stoi(this->ctx_.device_id));
  const Tc *dy = outputs[0]->get_grad_pointer<Tc>(this->ctx_);
  Tc *dx = inputs[0]->cast_grad_and_get_pointer<Tc>(this->ctx_, !accum[0]);
  const Size_t size = inputs[0]->size();
  NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_reduce_sum_backward, size, dx, dy);
}
}
