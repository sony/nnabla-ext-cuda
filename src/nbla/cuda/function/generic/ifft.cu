#include "hip/hip_runtime.h"
// Copyright 2018,2019,2020,2021 Sony Corporation.
// Copyright 2022 Sony Group Corporation.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <nbla/array.hpp>
#include <nbla/cuda/array/cuda_array.hpp>
#include <nbla/cuda/common.hpp>
#include <nbla/cuda/function/ifft.hpp>
#include <nbla/cuda/function/utils/fft.cuh>
#include <nbla/variable.hpp>

namespace nbla {

template <typename T> IFFTCuda<T>::~IFFTCuda() {
  NBLA_CUFFT_CHECK(hipfftDestroy(plan_forward_));
  NBLA_CUFFT_CHECK(hipfftDestroy(plan_backward_));
}

template <typename T>
void IFFTCuda<T>::setup_impl(const Variables &inputs,
                             const Variables &outputs) {
  cuda_set_device(this->device_);
  IFFT<T>::setup_impl(inputs, outputs);

  // Compute scale and store the original shape (i.e, n)
  Shape_t oshape(outputs[0]->shape());
  Size_t base_axis_output = oshape.size() - 1 - this->signal_ndim_;
  signal_size_ = 1;
  n_.clear();
  for (int i = 0; i < this->signal_ndim_; i++) {
    signal_size_ *= oshape[base_axis_output + i];
    n_.push_back(oshape[base_axis_output + i]);
  }
}

template <typename T>
void IFFTCuda<T>::forward_impl(const Variables &inputs,
                               const Variables &outputs) {
  cuda_set_device(this->device_);

  // IFFT
  const Tcu *x = inputs[0]->get_data_pointer<Tcu>(this->ctx_);
  Tcu *y = outputs[0]->cast_data_and_get_pointer<Tcu>(this->ctx_, true);
  exec_cufft<Tcu>(this->ctx_, x, y, inputs[0]->shape(), outputs[0]->shape(),
                  plan_forward_, true, true, HIPFFT_BACKWARD, this->n_,
                  this->signal_ndim_);

  // Normalize
  const Size_t size = outputs[0]->size();
  if (this->normalized_) {
    const float scale = 1.f / std::sqrt(this->signal_size_);
    NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_normalize_cufft_result, size, scale,
                                   y);
  } else {
    const float scale = 1.f / this->signal_size_;
    NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_normalize_cufft_result, size, scale,
                                   y);
  }
}

template <typename T>
void IFFTCuda<T>::backward_impl(const Variables &inputs,
                                const Variables &outputs,
                                const vector<bool> &propagate_down,
                                const vector<bool> &accum) {
  if (!(propagate_down[0])) {
    return;
  }
  cuda_set_device(this->device_);
  const Size_t size = inputs[0]->size();

  if (accum[0]) {
    // Create tmp array
    NdArrayPtr ndarray = make_shared<NdArray>(inputs[0]->shape());

    // FFT
    const Tcu *dy = outputs[0]->get_grad_pointer<Tcu>(this->ctx_);
    Tcu *tmp_buff = ndarray->cast(get_dtype<Tcu>(), this->ctx_)->pointer<Tcu>();
    exec_cufft<Tcu>(this->ctx_, dy, tmp_buff, outputs[0]->shape(),
                    inputs[0]->shape(), plan_backward_, true, true,
                    HIPFFT_FORWARD, this->n_, this->signal_ndim_);

    // Normalize
    const Size_t size = inputs[0]->size();
    if (this->normalized_) {
      const float scale = 1.f / std::sqrt(this->signal_size_);
      NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_normalize_cufft_result, size, scale,
                                     tmp_buff);
    } else {
      const float scale = 1.f / this->signal_size_;
      NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_normalize_cufft_result, size, scale,
                                     tmp_buff);
    }

    // Accumulation
    Tcu *dx = inputs[0]->cast_grad_and_get_pointer<Tcu>(this->ctx_, !accum[0]);
    NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_add_cufft_result, size, tmp_buff, dx);

  } else {
    // FFT
    const Tcu *dy = outputs[0]->get_grad_pointer<Tcu>(this->ctx_);
    Tcu *dx = inputs[0]->cast_grad_and_get_pointer<Tcu>(this->ctx_, !accum[0]);
    exec_cufft<Tcu>(this->ctx_, dy, dx, outputs[0]->shape(), inputs[0]->shape(),
                    plan_backward_, true, true, HIPFFT_FORWARD, this->n_,
                    this->signal_ndim_);
    // Normalize
    const Size_t size = inputs[0]->size();
    if (this->normalized_) {
      const float scale = 1.f / std::sqrt(this->signal_size_);
      NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_normalize_cufft_result, size, scale,
                                     dx);
    } else {
      const float scale = 1.f / this->signal_size_;
      NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_normalize_cufft_result, size, scale,
                                     dx);
    }
  }
}
}
