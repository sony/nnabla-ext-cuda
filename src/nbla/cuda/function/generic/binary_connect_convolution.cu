// Copyright 2018,2019,2020,2021 Sony Corporation.
// Copyright 2022 Sony Group Corporation.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <nbla/array.hpp>
#include <nbla/cuda/common.hpp>
#include <nbla/cuda/function/binary_connect_convolution.hpp>
#include <nbla/cuda/math.hpp>
#include <nbla/variable.hpp>

namespace nbla {

template <typename T>
void BinaryConnectConvolutionCuda<T>::setup_impl(const Variables &inputs,
                                                 const Variables &outputs) {
  BinaryConnectConvolution<T>::setup_impl(inputs, outputs);
}

template <typename T>
void BinaryConnectConvolutionCuda<T>::forward_impl(const Variables &inputs,
                                                   const Variables &outputs) {
  BinaryConnectConvolution<T>::forward_impl(inputs, outputs);
}

template <typename T>
void BinaryConnectConvolutionCuda<T>::backward_impl(
    const Variables &inputs, const Variables &outputs,
    const vector<bool> &propagate_down, const vector<bool> &accum) {
  BinaryConnectConvolution<T>::backward_impl(inputs, outputs, propagate_down,
                                             accum);
}
}
