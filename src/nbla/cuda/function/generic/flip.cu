#include "hip/hip_runtime.h"
// Copyright (c) 2017 Sony Corporation. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////
/* UNDER REVIEW.

   NOTE: hipMemcpy and kernel execution bat setup_impl.
*/
////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////

#include <nbla/array.hpp>
#include <nbla/cuda/common.hpp>
#include <nbla/cuda/function/flip.hpp>
#include <nbla/cuda/math.hpp>
#include <nbla/variable.hpp>

namespace nbla {

__global__ void kernel_flip_create_table(const int num, const int dim,
                                         int *addr_table_buf,
                                         const int *shape_info_buf) {
  NBLA_CUDA_KERNEL_LOOP(idx, num) {
    int addr = 0;
    for (int id = 0; id < dim; id++) {
      const int shape_info_offset = id * 3;
      const int o = (idx / shape_info_buf[shape_info_offset + 1]) // stride
                    % shape_info_buf[shape_info_offset];          // shape
      const int i = shape_info_buf[shape_info_offset + 2] ?       // flip
                        shape_info_buf[shape_info_offset] - 1 - o
                                                          : o;
      addr += i * shape_info_buf[shape_info_offset + 1]; // stride
    }
    addr_table_buf[idx] = addr;
  }
}

template <typename T>
void FlipCuda<T>::setup_impl(const Variables &inputs,
                             const Variables &outputs) {
  Flip<T>::setup_impl(inputs, outputs);

  // Prepare address table
  const Shape_t shape = outputs[0]->shape();
  const Shape_t stride = outputs[0]->strides();
  size_t size = outputs[0]->size();
  this->addr_table_.reshape(shape, true);
  const int shape_info_size = shape.size() * 3;
  // shape, stride, flip
  int *shape_info = new int[shape_info_size];
  for (int i = 0; i < shape.size(); i++) {
    shape_info[i * 3] = shape[i];
    shape_info[i * 3 + 1] = stride[i];
    auto itr = std::find(this->axes_.begin(), this->axes_.end(), i);
    shape_info[i * 3 + 2] = itr != this->axes_.end();
  }
  Shape_t shape_info_shape;
  shape_info_shape.push_back(shape_info_size);
  Variable shape_info_variable;
  shape_info_variable.reshape(shape_info_shape, true);
  int *shape_info_buf =
      shape_info_variable.cast_data_and_get_pointer<int>(this->ctx_, true);
  hipMemcpy(shape_info_buf, shape_info, sizeof(int) * shape_info_size,
             hipMemcpyHostToDevice);
  delete[] shape_info;
  Variable *addr_table_ = &this->addr_table_;
  int *addr_table_buf =
      addr_table_->cast_data_and_get_pointer<int>(this->ctx_, true);
  NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_flip_create_table, size, shape.size(),
                                 addr_table_buf, shape_info_buf);
}

template <typename T>
__global__ void kernel_flip_forward(const int num, T *y, const T *x,
                                    const int *addr_table_buf) {
  NBLA_CUDA_KERNEL_LOOP(idx, num) { y[idx] = x[addr_table_buf[idx]]; }
}

template <typename T>
void FlipCuda<T>::forward_impl(const Variables &inputs,
                               const Variables &outputs) {
  cuda_set_device(std::stoi(this->ctx_.device_id));
  const Tc *x = inputs[0]->get_data_pointer<Tc>(this->ctx_);
  const int *addr_table_buf =
      this->addr_table_.template get_data_pointer<int>(this->ctx_);
  Tc *y = outputs[0]->cast_data_and_get_pointer<Tc>(this->ctx_, true);
  size_t size = outputs[0]->size();
  NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_flip_forward, size, y, x,
                                 addr_table_buf);
}

template <typename T, bool accum>
__global__ void kernel_flip_backward(const int num, T *dx, const T *dy,
                                     const int *addr_table_buf) {
  NBLA_CUDA_KERNEL_LOOP(idx, num) {
    T &ref = dx[addr_table_buf[idx]];
    ref = (accum ? ref : (T)0) + dy[idx];
  }
}

template <typename T>
void FlipCuda<T>::backward_impl(const Variables &inputs,
                                const Variables &outputs,
                                const vector<bool> &propagate_down,
                                const vector<bool> &accum) {
  if (!propagate_down[0]) {
    return;
  }
  cuda_set_device(std::stoi(this->ctx_.device_id));
  Tc *dx = inputs[0]->cast_grad_and_get_pointer<Tc>(this->ctx_, !accum[0]);
  const int *addr_table_buf =
      this->addr_table_.template get_data_pointer<int>(this->ctx_);
  const Tc *dy = outputs[0]->get_grad_pointer<Tc>(this->ctx_);
  size_t size = outputs[0]->size();
  if (accum[0]) {
    NBLA_CUDA_LAUNCH_KERNEL_SIMPLE((kernel_flip_backward<Tc, true>), size, dx,
                                   dy, addr_table_buf);
  } else {
    NBLA_CUDA_LAUNCH_KERNEL_SIMPLE((kernel_flip_backward<Tc, false>), size, dx,
                                   dy, addr_table_buf);
  }
}
}
